#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<iostream>
#include<time.h>
#include<chrono>
#include<algorithm>

#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include <hipsparse.h>

#include "../utils/graph.h"
#include "../utils/io.h"

//#include<mkl.h>
//#include<mkl_solvers_ee.h>
//#include<mkl_spblas.h>
//#include<mkl_feast_evcount.h>

void printCsr(
    int m,
    int n,
    int nnz,
    const hipsparseMatDescr_t descrA,
    const float *csrValA,
    const int *csrRowPtrA,
    const int *csrColIndA,
    const char* name)
{
    const int base = (hipsparseGetMatIndexBase(descrA) != HIPSPARSE_INDEX_BASE_ONE)? 0:1 ;

    printf("matrix %s is %d-by-%d, nnz=%d, base=%d, output base-1\n", name, m, n, nnz, base);
    for(int row = 0 ; row < m ; row++){
        const int start = csrRowPtrA[row  ] - base;
        const int end   = csrRowPtrA[row+1] - base;
        for(int colidx = start ; colidx < end ; colidx++){
            const int col = csrColIndA[colidx] - base;
            const float Areg = csrValA[colidx];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}
__global__ void compute_d(double* deg, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= size) return;

	//id = id * size + id;
	
	/* Make assumption here that graph is  	*/
	/* connected and every node has degree 	*/
        /* atleast 1. 		       		*/

	deg[id] = sqrt(1/deg[id]); 
}

__global__ void compute_s(float* S, float* X, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] += X[id]; 
}

__global__ void transform_s(double* S, int volume, int window_size, int b, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = (S[id] * float(volume))/ ((float) window_size * (float) b); 
}

__global__ void prune_m(double* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;

	if(M[id] > 1+1e-10)
		M[id] = M[id];
	else
		M[id] = 0;	
}


__global__ void transform_m(double* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	M[id] =log(M[id]);
}

__global__ void sqrt_si(float* S, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = sqrt((float) S[id]);
}

void print_matrix(double* S, int size){
	std::cout<<std::endl<<std::endl;
	for(int i=0;i<size;i++){
		for(int j=0;j<size;j++){
			std::cout<<S[i*size + j]<<" ";
		}
		std::cout<<std::endl;
	}

}


int main ( void ){
	/**************
 	* NetMF small *
	**************/

	/* Settings */
	int window_size = 3;
	int dimension = 2;
	int b = 1;

	/* CuSparse housekeeping */
	hipsparseHandle_t cusparse_handle;    
	hipsparseCreate(&cusparse_handle);	


	/* Load graph */
        log("Reading data from file");

	Graph g =  read_graph("../data/test/small_test.csv","edgelist");

	log("Printing adj matrix");
	print_matrix(g.adj, g.size);	
	
	log("Printing degree matrix");
	print_matrix(g.degree, g.size);	
	/* Convert graph to sparse */	
	// Create dense device array

	log("Creating dense device array");
	double *adj_device_dense;	
	double *degree_device_dense; 

	log("Allocating space for dense mat on device");
	hipMalloc(&adj_device_dense, g.size * g.size * sizeof(double)); 	
	hipMalloc(&degree_device_dense, g.size * g.size * sizeof(double)); 

	log("Copying host to device");	
	hipMemcpy(adj_device_dense, g.adj, g.size * g.size * sizeof(double), hipMemcpyHostToDevice);	
	hipMemcpy(degree_device_dense, g.degree, g.size * g.size * sizeof(double), hipMemcpyHostToDevice);

	log("Creating matrix descriptors");	
	hipsparseMatDescr_t adj_descr;
	hipsparseCreateMatDescr(&adj_descr);
	hipsparseSetMatType(adj_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(adj_descr, HIPSPARSE_INDEX_BASE_ZERO);
	
	log("Creating matrix descriptors");	
	hipsparseMatDescr_t degree_descr;
	hipsparseCreateMatDescr(&degree_descr);
	hipsparseSetMatType(degree_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(degree_descr, HIPSPARSE_INDEX_BASE_ZERO);

	csr adj_csr, degree_csr;

	adj_csr.nnz = 0;
	degree_csr.nnz = 0;

	adj_csr.lda = g.size;
	degree_csr.lda = g.size;
	
	log("Computing nnzPerVector");	
	hipMalloc(&adj_csr.d_nnzPerVector, g.size * sizeof(double));
	hipsparseDnnz(cusparse_handle, HIPSPARSE_DIRECTION_ROW, g.size, g.size, adj_descr, adj_device_dense, adj_csr.lda, adj_csr.d_nnzPerVector, &adj_csr.nnz);

	hipMalloc(&degree_csr.d_nnzPerVector, g.size * sizeof(double));
	hipsparseDnnz(cusparse_handle, HIPSPARSE_DIRECTION_ROW, g.size, g.size, degree_descr, degree_device_dense, degree_csr.lda, degree_csr.d_nnzPerVector, &degree_csr.nnz);


	log("Computing nnzPerVector host");	
	adj_csr.h_nnzPerVector = (int *)malloc(g.size * sizeof(int));
	hipMemcpy(adj_csr.h_nnzPerVector, adj_csr.d_nnzPerVector, g.size * sizeof(int), hipMemcpyDeviceToHost);

	degree_csr.h_nnzPerVector = (int *)malloc(g.size * sizeof(int));
	hipMemcpy(degree_csr.h_nnzPerVector, degree_csr.d_nnzPerVector, g.size * sizeof(int), hipMemcpyDeviceToHost);

    	printf("Number of nonzero elements in dense adjacency matrix = %i\n\n", adj_csr.nnz);
    	for (int i = 0; i < g.size; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, adj_csr.h_nnzPerVector[i]);
    	printf("\n");

    	printf("Number of nonzero elements in dense degree matrix = %i\n\n", degree_csr.nnz);
    	for (int i = 0; i < g.size; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, degree_csr.h_nnzPerVector[i]);
    	printf("\n");

	log("Allocating host side sparse mat");
	adj_csr.h_values = (double *)malloc(adj_csr.nnz * sizeof(double));
	adj_csr.h_rowIndices = (int *)malloc((g.size+1) * sizeof(int));
	adj_csr.h_colIndices = (int *)malloc(adj_csr.nnz * sizeof(int));	
	
	log("Allocating host side sparse mat");
	degree_csr.h_values = (double *)malloc(degree_csr.nnz * sizeof(double));
	degree_csr.h_rowIndices = (int *)malloc((g.size+1) * sizeof(int));
	degree_csr.h_colIndices = (int *)malloc(degree_csr.nnz * sizeof(int));	

	log("Allocating device side sparse mat");
	hipMalloc(&adj_csr.d_values, adj_csr.nnz * sizeof(double));
	hipMalloc(&adj_csr.d_rowIndices, (g.size + 1) * sizeof(int));
	hipMalloc(&adj_csr.d_colIndices, adj_csr.nnz * sizeof(int));
	
	log("Allocating device side sparse mat");
	hipMalloc(&degree_csr.d_values, degree_csr.nnz * sizeof(double));
	hipMalloc(&degree_csr.d_rowIndices, (g.size + 1) * sizeof(int));
	hipMalloc(&degree_csr.d_colIndices, degree_csr.nnz * sizeof(int));

	hipsparseDdense2csr(cusparse_handle, 
			g.size, g.size, 
			adj_descr,
		        adj_device_dense,	
			adj_csr.lda, 
			adj_csr.d_nnzPerVector, 
			adj_csr.d_values, adj_csr.d_rowIndices, adj_csr.d_colIndices); 
	hipsparseDdense2csr(cusparse_handle, 
			g.size, g.size, 
			degree_descr, 
			degree_device_dense,
			degree_csr.lda, 
			degree_csr.d_nnzPerVector, 
			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices); 


	hipMemcpy(adj_csr.h_values, 
			adj_csr.d_values, 
			adj_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);

	hipMemcpy(adj_csr.h_rowIndices, 
			adj_csr.d_rowIndices, 
			(g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(adj_csr.h_colIndices, 
			adj_csr.d_colIndices, 
			adj_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(degree_csr.h_values, 
			degree_csr.d_values, 
			degree_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	
	hipMemcpy(degree_csr.h_rowIndices, 
			degree_csr.d_rowIndices, 
			(g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(degree_csr.h_colIndices,
		       	degree_csr.d_colIndices, 
			degree_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);

    	printf("\nOriginal adj matrix in CSR format\n\n");
    	for (int i = 0; i < adj_csr.nnz; ++i) printf("A[%i] = %f ", i, adj_csr.h_values[i]); printf("\n");

    	for (int i = 0; i < (g.size + 1); ++i) printf("RowIndices[%i] = %i \n", i, adj_csr.h_rowIndices[i]); printf("\n");

    	for (int i = 0; i < adj_csr.nnz; ++i) printf("ColIndices[%i] = %i \n", i, adj_csr.h_colIndices[i]);  

    	printf("\nOriginal degree matrix in CSR format\n\n");
    	for (int i = 0; i < degree_csr.nnz; ++i) printf("A[%i] = %f ", i, degree_csr.h_values[i]); printf("\n");

    	for (int i = 0; i < (g.size + 1); ++i) printf("RowIndices[%i] = %i \n", i, degree_csr.h_rowIndices[i]); printf("\n");

    	for (int i = 0; i < degree_csr.nnz; ++i) printf("ColIndices[%i] = %i \n", i, degree_csr.h_colIndices[i]);  

	/* CUDA housekeeping */
	float num_threads = 128;
	dim3 threads(num_threads);
	dim3 grids((int)ceil((float)g.size/num_threads));
	
	/* Compute D = D^{-1/2} */
	log("Computing normalized D");
	compute_d<<<grids, threads>>>(degree_csr.d_values, degree_csr.nnz);
	
	log("Computed normalized D");
	hipMemcpy(degree_csr.h_values, degree_csr.d_values, degree_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	for(int i=0;i<degree_csr.nnz;i++) std::cout<<degree_csr.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<degree_csr.nnz;i++) std::cout<<degree_csr.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<degree_csr.h_rowIndices[i]<< " "; std::cout<<"\n";

	/* Compute X = D^{-1/2}AD^{-1/2} */

	// Compute X_temp = D^{-1/2}A first	
	hipsparseMatDescr_t X_temp_descr;
	hipsparseCreateMatDescr(&X_temp_descr);
	hipsparseSetMatType(X_temp_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(X_temp_descr, HIPSPARSE_INDEX_BASE_ZERO);

	csr X_temp_csr;
	int baseX;
       	X_temp_csr.nnz = 0;
	int *nnzTotalDevHostPtr = &X_temp_csr.nnz;

	hipsparseStatus_t status;	
	hipError_t cuda_status;

	hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
	hipMalloc(&X_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int));
	status = hipsparseXcsrgemmNnz(cusparse_handle, 
				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				g.size, g.size, g.size,
				degree_descr, degree_csr.nnz,
				degree_csr.d_rowIndices, degree_csr.d_colIndices, 
				adj_descr, adj_csr.nnz,
				adj_csr.d_rowIndices, adj_csr.d_colIndices,
				X_temp_descr, X_temp_csr.d_rowIndices,
				nnzTotalDevHostPtr
				);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		std::cout << "nnz calculation failed" << std::endl;
		std::cout << "status = " << status << std::endl;
		exit(0);
	}		
	std::cout<<"Value of NNZ: "<<*nnzTotalDevHostPtr<<std::endl;

	if(NULL != nnzTotalDevHostPtr){
		X_temp_csr.nnz = *nnzTotalDevHostPtr;
	}else{
		hipMemcpy(&X_temp_csr.nnz, X_temp_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseX, X_temp_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
		X_temp_csr.nnz -= baseX;
	}

	hipMalloc(&X_temp_csr.d_colIndices, X_temp_csr.nnz * sizeof(int));
	hipMalloc(&X_temp_csr.d_values, X_temp_csr.nnz * sizeof(double));

	X_temp_csr.h_colIndices = (int *) malloc(X_temp_csr.nnz * sizeof(double));
	X_temp_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(double));
	X_temp_csr.h_values = (double *) malloc(X_temp_csr.nnz * sizeof(double));

	hipsparseDcsrgemm(cusparse_handle, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			g.size, g.size, g.size,
			degree_descr, degree_csr.nnz,
			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices,
			adj_descr, adj_csr.nnz,
			adj_csr.d_values, adj_csr.d_rowIndices, adj_csr.d_colIndices,
			X_temp_descr, 
			X_temp_csr.d_values, X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices);
	hipDeviceSynchronize();

	cuda_status = hipMemcpy(X_temp_csr.h_values, X_temp_csr.d_values, X_temp_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	std::cout<<cuda_status;
	std::cout<<cuda_status;
	cuda_status = hipMemcpy(X_temp_csr.h_rowIndices, X_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
	std::cout<<cuda_status;
	std::cout<<cuda_status;
	cuda_status = hipMemcpy(X_temp_csr.h_colIndices, X_temp_csr.d_colIndices, X_temp_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);

	std::cout<<cuda_status;

	log("Print X_temp");
	for(int i=0;i<X_temp_csr.nnz;i++) std::cout<<X_temp_csr.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<X_temp_csr.nnz;i++) std::cout<<X_temp_csr.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<X_temp_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
	
	// Compute X = X_tempD^{-1/2} second
	hipsparseMatDescr_t X_descr;
	hipsparseCreateMatDescr(&X_descr);
	hipsparseSetMatType(X_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(X_descr, HIPSPARSE_INDEX_BASE_ZERO);

	csr X_csr;
       	X_csr.nnz = 0;
	nnzTotalDevHostPtr = &X_csr.nnz;

	hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
	hipMalloc(&X_csr.d_rowIndices, (g.size + 1) * sizeof(int));
	status = hipsparseXcsrgemmNnz(cusparse_handle, 
				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				g.size, g.size, g.size,
				X_temp_descr, X_temp_csr.nnz,
				X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices, 
				degree_descr, degree_csr.nnz,
				degree_csr.d_rowIndices, degree_csr.d_colIndices,
				X_descr, X_csr.d_rowIndices,
				nnzTotalDevHostPtr
				);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		std::cout << "nnz calculation failed" << std::endl;
		std::cout << "status = " << status << std::endl;
		exit(0);
	}		
	std::cout<<"Value of NNZ: "<<*nnzTotalDevHostPtr<<std::endl;

	if(NULL != nnzTotalDevHostPtr){
		X_csr.nnz = *nnzTotalDevHostPtr;
	}else{
		hipMemcpy(&X_csr.nnz, X_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseX, X_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
		X_csr.nnz -= baseX;
	}

	hipMalloc(&X_csr.d_colIndices, X_csr.nnz * sizeof(int));
	hipMalloc(&X_csr.h_rowIndices, (g.size + 1) * sizeof(int));
	hipMalloc(&X_csr.d_values, X_csr.nnz * sizeof(double));

	X_csr.h_colIndices = (int *) malloc(X_csr.nnz * sizeof(double));
	X_csr.h_rowIndices = (int *) malloc(X_csr.nnz * sizeof(double));
	X_csr.h_values = (double *) malloc(X_csr.nnz * sizeof(double));

	hipsparseDcsrgemm(cusparse_handle, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			g.size, g.size, g.size,
			X_temp_descr, X_temp_csr.nnz,
			X_temp_csr.d_values, X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices,
			degree_descr, degree_csr.nnz,
			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices,
			X_descr, 
			X_csr.d_values, X_csr.d_rowIndices, X_csr.d_colIndices);
	hipDeviceSynchronize();

	cuda_status = hipMemcpy(X_csr.h_values, X_csr.d_values, X_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	cuda_status = hipMemcpy(X_csr.h_rowIndices, X_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
	cuda_status = hipMemcpy(X_csr.h_colIndices, X_csr.d_colIndices, X_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);

	std::cout<<cuda_status;

	log("Print X");
	for(int i=0;i<X_csr.nnz;i++) std::cout<<X_csr.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<X_csr.nnz;i++) std::cout<<X_csr.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<X_csr.h_rowIndices[i]<< " "; std::cout<<"\n";


	 /* Compute S = sum(X^{0}....X^{window_size}) */

	csr S_csr, temp_csr, temp1_csr, S_temp_csr;
	S_csr.nnz = X_csr.nnz;
	temp_csr.nnz = X_csr.nnz;

	hipMalloc(&S_csr.d_values, S_csr.nnz * sizeof(double));
	hipMalloc(&S_csr.d_colIndices, S_csr.nnz * sizeof(int));
	hipMalloc(&S_csr.d_rowIndices, (g.size + 1) * sizeof(int));

	hipMalloc(&temp_csr.d_values, temp_csr.nnz * sizeof(double));
	hipMalloc(&temp_csr.d_colIndices, temp_csr.nnz * sizeof(int));
	hipMalloc(&temp_csr.d_rowIndices, (g.size + 1) * sizeof(int));

	log("Copying X to S");
	hipMemcpy(S_csr.d_values, X_csr.d_values, S_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(S_csr.d_colIndices, X_csr.d_colIndices, S_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(S_csr.d_rowIndices, X_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);

	log("Copying S to host");
	S_csr.h_values = (double *) malloc(S_csr.nnz * sizeof(double));
	S_csr.h_colIndices = (int *) malloc(S_csr.nnz * sizeof(int));
	S_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(int));

	hipMemcpy(S_csr.h_values, S_csr.d_values, S_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(S_csr.h_colIndices, S_csr.d_colIndices, S_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(S_csr.h_rowIndices, S_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<S_csr.nnz;i++) std::cout<<S_csr.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<S_csr.nnz;i++) std::cout<<S_csr.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<S_csr.h_rowIndices[i]<< " "; std::cout<<"\n";

	log("Copying X to temp");
	hipMemcpy(temp_csr.d_values, X_csr.d_values, temp_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(temp_csr.d_colIndices, X_csr.d_colIndices, temp_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(temp_csr.d_rowIndices, X_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);

	log("Copying S to host");
	temp_csr.h_values = (double *) malloc(temp_csr.nnz * sizeof(double));
	temp_csr.h_colIndices = (int *) malloc(temp_csr.nnz * sizeof(int));
	temp_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(int));

	hipMemcpy(temp_csr.h_values, temp_csr.d_values, temp_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(temp_csr.h_colIndices, temp_csr.d_colIndices, temp_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(temp_csr.h_rowIndices, temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<temp_csr.nnz;i++) std::cout<<temp_csr.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<temp_csr.nnz;i++) std::cout<<temp_csr.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<temp_csr.h_rowIndices[i]<< " "; std::cout<<"\n";


	hipsparseMatDescr_t S_descr;
	hipsparseCreateMatDescr(&S_descr);
	hipsparseSetMatType(S_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(S_descr, HIPSPARSE_INDEX_BASE_ZERO);

	hipsparseMatDescr_t S_temp_descr;
	hipsparseCreateMatDescr(&S_temp_descr);
	hipsparseSetMatType(S_temp_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(S_temp_descr, HIPSPARSE_INDEX_BASE_ZERO);
	
	hipsparseMatDescr_t temp_descr;
	hipsparseCreateMatDescr(&temp_descr);
	hipsparseSetMatType(temp_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(temp_descr, HIPSPARSE_INDEX_BASE_ZERO);

	hipsparseMatDescr_t temp1_descr;
	hipsparseCreateMatDescr(&temp1_descr);
	hipsparseSetMatType(temp1_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(temp1_descr, HIPSPARSE_INDEX_BASE_ZERO);

	double alf = 1.00;
	double beta = 1.00;

	for(int i=2;i<=window_size;i++){
		std::cout<<"Computing X^"<<i<<std::endl;
		temp1_csr.nnz = 0;
		temp1_csr.d_rowIndices = NULL;
		temp1_csr.d_colIndices = NULL;
		temp1_csr.d_values = NULL;
		
		temp1_csr.h_rowIndices = NULL;
		temp1_csr.h_colIndices = NULL;
		temp1_csr.h_values = NULL;
		
		S_temp_csr.nnz = 0;
		S_temp_csr.d_rowIndices = NULL;
		S_temp_csr.d_colIndices = NULL;
		S_temp_csr.d_values = NULL;
		
		S_temp_csr.h_rowIndices = NULL;
		S_temp_csr.h_colIndices = NULL;
		S_temp_csr.h_values = NULL;

		nnzTotalDevHostPtr = &temp1_csr.nnz;

		hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
		hipMalloc(&temp1_csr.d_rowIndices, (g.size + 1) * sizeof(double));
		status = hipsparseXcsrgemmNnz(cusparse_handle, 
					HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
					g.size, g.size, g.size,
					temp_descr, temp_csr.nnz,
					temp_csr.d_rowIndices, temp_csr.d_colIndices,
					X_descr, X_csr.nnz,
					X_csr.d_rowIndices, X_csr.d_colIndices,
					temp1_descr, temp1_csr.d_rowIndices,
					nnzTotalDevHostPtr);

		if(status != 0){
			std::cout<<"Error encountered"<<std::endl;
			std::cout<<"Status: "<<status<<std::endl;
			exit(0);	

		}


		if(NULL!=nnzTotalDevHostPtr) temp1_csr.nnz = *nnzTotalDevHostPtr;
		else{
			hipMemcpy(&temp1_csr.nnz, temp1_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&baseX, temp1_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
			temp1_csr.nnz -= baseX;
		
		}

		std::cout<<"NNZ ="<<temp1_csr.nnz<<std::endl;

		hipMalloc(&temp1_csr.d_values, temp1_csr.nnz * sizeof(double));
		hipMalloc(&temp1_csr.d_colIndices, temp1_csr.nnz * sizeof(int));

		hipsparseDcsrgemm(cusparse_handle,
				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				g.size, g.size, g.size,
				temp_descr, temp_csr.nnz,
				temp_csr.d_values, temp_csr.d_rowIndices, temp_csr.d_colIndices,
				X_descr, X_csr.nnz,
				X_csr.d_values, X_csr.d_rowIndices, X_csr.d_colIndices,
				temp1_descr, 
				temp1_csr.d_values, temp1_csr.d_rowIndices, temp1_csr.d_colIndices);
	
		log("Printing intermediate result");					
		temp1_csr.h_values = (double *) malloc(temp1_csr.nnz * sizeof(double));
		temp1_csr.h_colIndices = (int *) malloc(temp1_csr.nnz * sizeof(int));
		temp1_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(int));

		hipMemcpy(temp1_csr.h_values, temp1_csr.d_values, temp1_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(temp1_csr.h_colIndices, temp1_csr.d_colIndices, temp1_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(temp1_csr.h_rowIndices, temp1_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);

		for(int i=0;i<temp1_csr.nnz;i++) std::cout<<temp1_csr.h_values[i]<< " "; std::cout<<"\n";
		for(int i=0;i<temp1_csr.nnz;i++) std::cout<<temp1_csr.h_colIndices[i]<< " "; std::cout<<"\n";
		for(int i=0;i<g.size + 1;i++) std::cout<<temp1_csr.h_rowIndices[i]<< " "; std::cout<<"\n";

		hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);

		S_temp_csr.nnz = 0;
		nnzTotalDevHostPtr = &S_temp_csr.nnz;
		hipMalloc(&S_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int));
		
		status = hipsparseXcsrgeamNnz(cusparse_handle,
					g.size, g.size,
					temp1_descr, temp1_csr.nnz, temp1_csr.d_rowIndices, temp1_csr.d_colIndices,
					S_descr,S_csr.nnz, S_csr.d_rowIndices, S_csr.d_colIndices,
					S_temp_descr, S_temp_csr.d_rowIndices, nnzTotalDevHostPtr);	

		if(status != 0){
			std::cout<<"Error encountered"<<std::endl;
			std::cout<<"Status: "<<status<<std::endl;
			exit(0);	

		}
		if(NULL!=nnzTotalDevHostPtr) S_temp_csr.nnz = *nnzTotalDevHostPtr;
		else{
			hipMemcpy(&S_temp_csr.nnz, S_temp_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&baseX, S_temp_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
			S_temp_csr.nnz -= baseX;
		}

		std::cout<<"SUM-NNZ"<<S_temp_csr.nnz<<std::endl;

		hipMalloc(&S_temp_csr.d_colIndices, sizeof(int) * S_temp_csr.nnz);
		hipMalloc(&S_temp_csr.d_values, sizeof(double) * S_temp_csr.nnz);

		hipsparseDcsrgeam(cusparse_handle, g.size, g.size,
				&alf,
				temp1_descr, temp1_csr.nnz,
				temp1_csr.d_values, temp1_csr.d_rowIndices, temp1_csr.d_colIndices,
				&beta,
				S_descr, S_csr.nnz,
				S_csr.d_values, S_csr.d_rowIndices, S_csr.d_colIndices,
				S_temp_descr,
				S_temp_csr.d_values, S_temp_csr.d_rowIndices, S_temp_csr.d_colIndices);

		log("Printing intermediate sum");
		S_temp_csr.h_values = (double *) malloc(sizeof(double) * S_temp_csr.nnz);
		S_temp_csr.h_rowIndices = (int *) malloc(sizeof(int) * (g.size + 1));
		S_temp_csr.h_colIndices = (int *) malloc(sizeof(int) * S_temp_csr.nnz);

		hipMemcpy(S_temp_csr.h_values, S_temp_csr.d_values, S_temp_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(S_temp_csr.h_rowIndices, S_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(S_temp_csr.h_colIndices, S_temp_csr.d_colIndices, S_temp_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);

		for(int i=0;i<S_temp_csr.nnz;i++) std::cout<<S_temp_csr.h_values[i]<< " "; std::cout<<"\n";
		for(int i=0;i<S_temp_csr.nnz;i++) std::cout<<S_temp_csr.h_colIndices[i]<< " "; std::cout<<"\n";
		for(int i=0;i<g.size + 1;i++) std::cout<<S_temp_csr.h_rowIndices[i]<< " "; std::cout<<"\n";

		S_csr.nnz = S_temp_csr.nnz;
		S_csr.d_values = NULL;
		S_csr.d_rowIndices = NULL;
		S_csr.d_colIndices = NULL;
		
		hipMalloc(&S_csr.d_values, S_csr.nnz * sizeof(double)); 
		hipMalloc(&S_csr.d_rowIndices, (g.size + 1) * sizeof(int)); 
		hipMalloc(&S_csr.d_colIndices, S_csr.nnz * sizeof(int)); 

		hipMemcpy(S_csr.d_values, S_temp_csr.d_values, S_temp_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
		hipMemcpy(S_csr.d_rowIndices, S_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(S_csr.d_colIndices, S_temp_csr.d_colIndices, S_temp_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);


		temp_csr.nnz = temp1_csr.nnz;
		temp_csr.d_values = NULL;
		temp_csr.d_rowIndices = NULL;
		temp_csr.d_colIndices = NULL;

		hipMalloc(&temp_csr.d_values, temp_csr.nnz * sizeof(double)); 
		hipMalloc(&temp_csr.d_rowIndices, (g.size + 1) * sizeof(int)); 
		hipMalloc(&temp_csr.d_colIndices, temp_csr.nnz * sizeof(int)); 

		hipMemcpy(temp_csr.d_values, temp1_csr.d_values, temp_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
		hipMemcpy(temp_csr.d_rowIndices, temp1_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(temp_csr.d_colIndices, temp1_csr.d_colIndices, temp_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);
			
	}

        /* Compute S = S * (vol / (window_size * b)) */

	transform_s<<<grids, threads>>>(S_csr.d_values, g.volume, window_size, b, S_csr.nnz);

	S_csr.h_values = (double *)malloc(sizeof(double) * S_csr.nnz);

	hipMemcpy(S_csr.h_values, S_csr.d_values, S_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);

	log("Transformed S values");
	for(int i=0;i<S_csr.nnz;i++) std::cout<<S_csr.h_values[i]<< " "; std::cout<<"\n";

	log("Computing M");


        /* Compute M = D^{-1/2} * S * D^{-1/2} */

	// Compute X_temp = D^{-1/2} * S first	
       	X_temp_csr.nnz = 0;
	nnzTotalDevHostPtr = &X_temp_csr.nnz;

	hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
	hipMalloc(&X_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int));
	status = hipsparseXcsrgemmNnz(cusparse_handle, 
				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				g.size, g.size, g.size,
				degree_descr, degree_csr.nnz,
				degree_csr.d_rowIndices, degree_csr.d_colIndices, 
				S_descr, S_csr.nnz,
				S_csr.d_rowIndices, S_csr.d_colIndices,
				X_temp_descr, X_temp_csr.d_rowIndices,
				nnzTotalDevHostPtr
				);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		std::cout << "nnz calculation failed" << std::endl;
		std::cout << "status = " << status << std::endl;
		exit(0);
	}		
	std::cout<<"Value of NNZ: "<<*nnzTotalDevHostPtr<<std::endl;

	if(NULL != nnzTotalDevHostPtr){
		X_temp_csr.nnz = *nnzTotalDevHostPtr;
	}else{
		hipMemcpy(&X_temp_csr.nnz, X_temp_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseX, X_temp_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
		X_temp_csr.nnz -= baseX;
	}

	hipMalloc(&X_temp_csr.d_colIndices, X_temp_csr.nnz * sizeof(int));
	hipMalloc(&X_temp_csr.d_values, X_temp_csr.nnz * sizeof(double));

	X_temp_csr.h_colIndices = (int *) malloc(X_temp_csr.nnz * sizeof(double));
	X_temp_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(double));
	X_temp_csr.h_values = (double *) malloc(X_temp_csr.nnz * sizeof(double));

	hipsparseDcsrgemm(cusparse_handle, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			g.size, g.size, g.size,
			degree_descr, degree_csr.nnz,
			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices,
			S_descr, S_csr.nnz,
			S_csr.d_values, S_csr.d_rowIndices, S_csr.d_colIndices,
			X_temp_descr, 
			X_temp_csr.d_values, X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices);
	hipDeviceSynchronize();

	cuda_status = hipMemcpy(X_temp_csr.h_values, X_temp_csr.d_values, X_temp_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	cuda_status = hipMemcpy(X_temp_csr.h_rowIndices, X_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
	cuda_status = hipMemcpy(X_temp_csr.h_colIndices, X_temp_csr.d_colIndices, X_temp_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);

	log("Print X_temp");
	for(int i=0;i<X_temp_csr.nnz;i++) std::cout<<X_temp_csr.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<X_temp_csr.nnz;i++) std::cout<<X_temp_csr.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<X_temp_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
	
	// Compute M = X_temp * D^{-1/2} second
	hipsparseMatDescr_t M_descr;
	hipsparseCreateMatDescr(&M_descr);
	hipsparseSetMatType(M_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(M_descr, HIPSPARSE_INDEX_BASE_ZERO);

	csr M_csr;
       	M_csr.nnz = 0;
	nnzTotalDevHostPtr = &M_csr.nnz;

	hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
	hipMalloc(&M_csr.d_rowIndices, (g.size + 1) * sizeof(int));
	status = hipsparseXcsrgemmNnz(cusparse_handle, 
				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				g.size, g.size, g.size,
				X_temp_descr, X_temp_csr.nnz,
				X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices, 
				degree_descr, degree_csr.nnz,
				degree_csr.d_rowIndices, degree_csr.d_colIndices,
				M_descr, M_csr.d_rowIndices,
				nnzTotalDevHostPtr
				);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		std::cout << "nnz calculation failed" << std::endl;
		std::cout << "status = " << status << std::endl;
		exit(0);
	}		
	std::cout<<"Value of NNZ: "<<*nnzTotalDevHostPtr<<std::endl;

	if(NULL != nnzTotalDevHostPtr){
		M_csr.nnz = *nnzTotalDevHostPtr;
	}else{
		hipMemcpy(&M_csr.nnz, M_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseX, M_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
		M_csr.nnz -= baseX;
	}

	hipMalloc(&M_csr.d_colIndices, M_csr.nnz * sizeof(int));
	hipMalloc(&M_csr.h_rowIndices, (g.size + 1) * sizeof(int));
	hipMalloc(&M_csr.d_values, M_csr.nnz * sizeof(double));

	M_csr.h_colIndices = (int *) malloc(M_csr.nnz * sizeof(double));
	M_csr.h_rowIndices = (int *) malloc(M_csr.nnz * sizeof(double));
	M_csr.h_values = (double *) malloc(M_csr.nnz * sizeof(double));

	hipsparseDcsrgemm(cusparse_handle, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			g.size, g.size, g.size,
			X_temp_descr, X_temp_csr.nnz,
			X_temp_csr.d_values, X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices,
			degree_descr, degree_csr.nnz,
			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices,
			M_descr, 
			M_csr.d_values, M_csr.d_rowIndices, M_csr.d_colIndices);
	hipDeviceSynchronize();

	cuda_status = hipMemcpy(M_csr.h_values, M_csr.d_values, M_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	cuda_status = hipMemcpy(M_csr.h_rowIndices, M_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
	cuda_status = hipMemcpy(M_csr.h_colIndices, M_csr.d_colIndices, M_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);

	std::cout<<cuda_status;

	log("Print M");
	for(int i=0;i<M_csr.nnz;i++) std::cout<<M_csr.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<M_csr.nnz;i++) std::cout<<M_csr.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<M_csr.h_rowIndices[i]<< " "; std::cout<<"\n";

	log("Transforming M");

	prune_m<<<grids,threads>>>(M_csr.d_values, M_csr.nnz);
	hipMemcpy(M_csr.h_values, M_csr.d_values, M_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
	for(int i=0;i<M_csr.nnz;i++) std::cout<<M_csr.h_values[i]<< " "; std::cout<<"\n";
	hipDeviceSynchronize();



	double threshold = 1.00;
	log("Setting threshold");
	std::cout<<"Threshold:"<<threshold;
	csr filtered_M;

	hipMalloc(&filtered_M.d_rowIndices, sizeof(int) * (g.size + 1));
	size_t lworkInBytes = 0;
	char *d_work=NULL;
	hipsparseDpruneCsr2csr_bufferSizeExt(cusparse_handle,
        					g.size,g.size,
        					M_csr.nnz,M_descr,
        					M_csr.d_values,M_csr.d_rowIndices,M_csr.d_colIndices,
        					&threshold,
        					M_descr,
        					filtered_M.d_values, filtered_M.d_rowIndices, filtered_M.d_colIndices,
        					&lworkInBytes);	

    	printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);
	hipMalloc((void**)&d_work, lworkInBytes);

	hipsparseDpruneCsr2csrNnz(cusparse_handle,
					g.size,g.size,
        				M_csr.nnz,M_descr,
  					M_csr.d_values,M_csr.d_rowIndices,M_csr.d_colIndices,
        				&threshold,
        				M_descr,
        				filtered_M.d_rowIndices, &filtered_M.nnz, /* host */
        				d_work);

	printf("nnzC = %d\n", filtered_M.nnz);
    	if (0 == filtered_M.nnz ){
        	printf("C is empty \n");
        	return 0;
    	}


	hipMalloc(&filtered_M.d_colIndices, sizeof(int) * filtered_M.nnz);
	hipMalloc(&filtered_M.d_values, sizeof(double) * filtered_M.nnz);

	hipsparseDpruneCsr2csr(cusparse_handle,
   				g.size,g.size,
        			M_csr.nnz, M_descr,
        			M_csr.d_values, M_csr.d_rowIndices, M_csr.d_colIndices,
        			&threshold,
        			M_descr,
        			filtered_M.d_values,filtered_M.d_rowIndices, filtered_M.d_colIndices,
        			d_work);


	log("Printing pruned M");
	filtered_M.h_values = (double *) malloc(sizeof(double) * filtered_M.nnz);
	filtered_M.h_colIndices = (int *) malloc(sizeof(int) * filtered_M.nnz);
	filtered_M.h_rowIndices = (int *) malloc(sizeof(int) * (g.size + 1));

	hipMemcpy(filtered_M.h_values, filtered_M.d_values, sizeof(double) * filtered_M.nnz, hipMemcpyDeviceToHost);
	hipMemcpy(filtered_M.h_colIndices, filtered_M.d_colIndices, sizeof(int) * filtered_M.nnz, hipMemcpyDeviceToHost);
	hipMemcpy(filtered_M.h_rowIndices, filtered_M.d_rowIndices, sizeof(int) * (g.size + 1), hipMemcpyDeviceToHost);
	
	for(int i=0;i<filtered_M.nnz;i++) std::cout<<filtered_M.h_values[i]<< " "; std::cout<<"\n";
	for(int i=0;i<filtered_M.nnz;i++) std::cout<<filtered_M.h_colIndices[i]<< " "; std::cout<<"\n";
	for(int i=0;i<g.size + 1;i++) std::cout<<filtered_M.h_rowIndices[i]<< " "; std::cout<<"\n";
	
	log("Printing log of M");
	transform_m<<<grids,threads>>>(filtered_M.d_values, filtered_M.nnz);

	hipMemcpy(filtered_M.h_values, filtered_M.d_values, sizeof(double) * filtered_M.nnz, hipMemcpyDeviceToHost);
	for(int i=0;i<filtered_M.nnz;i++) std::cout<<filtered_M.h_values[i]<< " "; std::cout<<"\n";

		

}
