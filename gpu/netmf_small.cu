#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<iostream>
#include<time.h>
#include<chrono>
#include<algorithm>

#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include <hipsparse.h>

#include "../utils/graph.h"
#include "../utils/io.h"

#include<mkl.h>
#include<mkl_solvers_ee.h>
#include<mkl_spblas.h>
//#include<mkl_feast_evcount.h>

#define DEBUG true


void print_csr(
    int m,
    int nnz,
    csr mat_csr,
    const char* name)
{
    printf("matrix %s is %d-by-%d, nnz=%d\n", name, m, m, nnz);
    std::cout<<"Values: "; for(int i=0;i<nnz;i++) std::cout<<mat_csr.h_values[i]<<" "; std::cout<<'\n';
    std::cout<<"Cols: "; for(int i=0;i<nnz;i++) std::cout<<mat_csr.h_colIndices[i]<<" "; std::cout<<'\n';
    std::cout<<"Rows: "; for(int i=0;i<m+1;i++) std::cout<<mat_csr.h_rowIndices[i]<<" "; std::cout<<'\n';
}
__global__ void compute_d(double* deg, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= size) return;

	//id = id * size + id;
	
	/* Make assumption here that graph is  	*/
	/* connected and every node has degree 	*/
        /* atleast 1. 		       		*/

	deg[id] = sqrt(1/deg[id]); 
}

__global__ void compute_s(float* S, float* X, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] += X[id]; 
}

__global__ void transform_s(double* S, int volume, int window_size, int b, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = (S[id] * float(volume))/ ((float) window_size * (float) b); 
}

__global__ void prune_m(double* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;

	if(M[id] > 1+1e-10)
		M[id] = M[id];
	else
		M[id] = 0;	
}


__global__ void transform_m(double* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	M[id] =log(M[id]);
}

__global__ void sqrt_si(float* S, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = sqrt((float) S[id]);
}

void print_matrix(double* S, int size){
	std::cout<<std::endl<<std::endl;
	for(int i=0;i<size;i++){
		for(int j=0;j<size;j++){
			std::cout<<S[i*size + j]<<" ";
		}
		std::cout<<std::endl;
	}

}


void allocate_csr_row(csr *csr_mat, int num_rows){
	csr_mat->h_rowIndices = (int *) malloc((num_rows+1) * sizeof(int));
	hipMalloc(&csr_mat->d_rowIndices, (num_rows+1) * sizeof(int));

}
void allocate_csr_col_val(csr *csr_mat, int nnz){
	csr_mat->h_values = (double *) malloc(nnz * sizeof(double));
	csr_mat->h_colIndices = (int *) malloc(nnz * sizeof(int));

	hipMalloc(&csr_mat->d_values, nnz*sizeof(double));
	hipMalloc(&csr_mat->d_colIndices, nnz * sizeof(int));

}

void allocate_csr(csr *csr_mat, int nnz, int num_rows){
	csr_mat->h_values = (double *) malloc(nnz * sizeof(double));
	csr_mat->h_colIndices = (int *) malloc(nnz * sizeof(int));
	csr_mat->h_rowIndices = (int *) malloc((num_rows+1) * sizeof(int));

	hipMalloc(&csr_mat->d_values, nnz*sizeof(double));
	hipMalloc(&csr_mat->d_colIndices, nnz * sizeof(int));
	hipMalloc(&csr_mat->d_rowIndices, (num_rows+1) * sizeof(int));

}


void device2host(csr *csr_mat, int nnz, int num_rows){
	hipMemcpy(csr_mat->h_values, csr_mat->d_values, 
			nnz * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(csr_mat->h_colIndices, csr_mat->d_colIndices, 
			nnz * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(csr_mat->h_rowIndices, csr_mat->d_rowIndices, 
			(num_rows + 1) * sizeof(int), hipMemcpyDeviceToHost);
}

void host2device(csr *csr_mat, int nnz, int num_rows){
	hipMemcpy(csr_mat->d_values, csr_mat->h_values, 
			nnz * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(csr_mat->d_colIndices, csr_mat->h_colIndices, 
			nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(csr_mat->d_rowIndices, csr_mat->h_rowIndices, 
			(num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
}

void multiply_csr(csr *A, csr *B, csr *C, int m, int n, int k, hipsparseHandle_t context,hipsparseMatDescr_t descr ){

	hipsparseStatus_t status;

	int base;
	int *nnzTotalDevHostPtr;
	C->nnz = 0;
	nnzTotalDevHostPtr = &C->nnz;

	hipsparseSetPointerMode(context, HIPSPARSE_POINTER_MODE_HOST);

	allocate_csr_row(C, m);
	hipMalloc(&C->d_rowIndices, (m+1) * sizeof(int));
	
	status = hipsparseXcsrgemmNnz(context, 
                      HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      m, n, k,
                      descr, A->nnz,
                      A->d_rowIndices, A->d_colIndices, 
                      descr, B->nnz,
                      B->d_rowIndices, B->d_colIndices,
                      descr, C->d_rowIndices,
                      nnzTotalDevHostPtr
                      );

	if(status!=HIPSPARSE_STATUS_SUCCESS){
		std::cout<<"Error occured in finding NNZ";
		std::cout<<"\n Status "<<status;
		exit(0);
	}

	if(NULL != nnzTotalDevHostPtr){
		C->nnz = *nnzTotalDevHostPtr;
	}else{
		hipMemcpy(&C->nnz, C->d_rowIndices + m, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&base, C->d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
		C->nnz -= base;
	}

	allocate_csr_col_val(C, C->nnz);	

	hipsparseDcsrgemm(context, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m,n,k,
			descr, A->nnz,
			A->d_values, A->d_rowIndices, A->d_colIndices,
			descr, B->nnz,
			B->d_values, B->d_rowIndices, B->d_colIndices,
			descr, 
			C->d_values, C->d_rowIndices, C->d_colIndices);

	hipDeviceSynchronize();
}


int main ( void ){
	/**************
 	* NetMF small *
	**************/

	/* Section 0: Preliminaries */

	/* Settings */
	int window_size = 3;
	int dimension = 128;
	int b = 1;

	/* Load graph */
        log("Reading data from file");
	
	Graph g =  read_graph("../data/test/small_test.csv","edgelist");

	log("Printing adj matrix");
	if(DEBUG)
		print_matrix(g.adj, g.size);	
	
	log("Printing degree matrix");
	if(DEBUG)
		print_matrix(g.degree, g.size);

	/* CUDA housekeeping */
	float num_threads = 128;
	dim3 threads(num_threads);
	dim3 grids((int)ceil((float)g.size/num_threads));

	/* CuSparse housekeeping */
	hipsparseHandle_t cusparse_handle;    
	hipsparseCreate(&cusparse_handle);	

	hipsparseMatDescr_t mat_descr;
	hipsparseCreateMatDescr(&mat_descr);
	hipsparseSetMatType(mat_descr, 
			HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(mat_descr, 
			HIPSPARSE_INDEX_BASE_ZERO);
	int LDA = g.size;


	/* Section 1. Convert graph to sparse */	

	/* Procedure 
	   1. Create dense adjacency and degree matrix on device
	   2. Allocate space for adjacency and degree matrix on device
	   3. Copy dense matrix from host to device
	   4. Create CSR data structure for both matrices
	   5. Compute nnz/row of dense matrix
	   6. Apply Dense2CSR
	 */

	/* Step 1: Create dense adjacency matrix and degree matrixx on device */
	log("Creating dense device array");
	double *adj_device_dense;	
	double *degree_device_dense; 

	/* Step 2: Allocate space for adjacency and degree matrix on device */
	log("Allocating space for dense mat on device");
	hipMalloc(&adj_device_dense, 
			g.size * g.size * sizeof(double)); 	
	hipMalloc(&degree_device_dense, 
			g.size * g.size * sizeof(double)); 

	/* Step 3: Copy dense matrix from host to device */
	log("Copying dense matrix from host to device");	
	hipMemcpy(adj_device_dense, 
			g.adj, 
			g.size * g.size * sizeof(double), 
			hipMemcpyHostToDevice);	
	hipMemcpy(degree_device_dense, 
			g.degree, 
			g.size * g.size * sizeof(double), 
			hipMemcpyHostToDevice);

	/* Step 4: Create CSR struct for both matrices */
	log("Converting dense matrix to CSR format");	
	csr adj_csr,    /* Variable to hold adjacency matrix in CSR format */
	    degree_csr; /* Variable to hold degree matrix in CSR format */

	adj_csr.nnz = 0; /* Initialize number of non zeros in adjacency matrix */
	degree_csr.nnz = 0; /* Initialize number of non zeros in degree matrix */

	/* Step 5: Compute nnz/row of dense matrix */	
	log("Computing nnzPerVector for A");

	hipMalloc(&adj_csr.d_nnzPerVector, 
			g.size * sizeof(int));
	hipsparseDnnz(cusparse_handle, 
			HIPSPARSE_DIRECTION_ROW, 
			g.size, g.size, 
			mat_descr, 
			adj_device_dense, LDA, 
			adj_csr.d_nnzPerVector, &adj_csr.nnz);
	adj_csr.h_nnzPerVector = (int *)malloc(g.size * sizeof(int));
	hipMemcpy(adj_csr.h_nnzPerVector, 
			adj_csr.d_nnzPerVector, 
			g.size * sizeof(int), 
			hipMemcpyDeviceToHost);
	
	if(DEBUG){
    		printf("Number of nonzero elements in dense adjacency matrix = %i\n", adj_csr.nnz);
    		
		for (int i = 0; i < g.size; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, adj_csr.h_nnzPerVector[i]);
	}

	log("Computing nnzPerVector for D");
	hipMalloc(&degree_csr.d_nnzPerVector, 
			g.size * sizeof(int));
	hipsparseDnnz(cusparse_handle, 
			HIPSPARSE_DIRECTION_ROW, 
			g.size, g.size, 
			mat_descr, 
			degree_device_dense, LDA, 
			degree_csr.d_nnzPerVector, &degree_csr.nnz);
	degree_csr.h_nnzPerVector = (int *)malloc(g.size * sizeof(int));
	hipMemcpy(degree_csr.h_nnzPerVector, 
			degree_csr.d_nnzPerVector, 
			g.size * sizeof(int), 
			hipMemcpyDeviceToHost);


	if(DEBUG){
    		printf("Number of nonzero elements in dense degree matrix = %i\n", degree_csr.nnz);
    		for (int i = 0; i < g.size; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, degree_csr.h_nnzPerVector[i]);
	}


	/* Step 6: Convert dense matrix to sparse matrices */
	allocate_csr(&adj_csr, adj_csr.nnz, g.size);
	hipsparseDdense2csr(cusparse_handle, 
			g.size, g.size, 
			mat_descr,
		        adj_device_dense,	
			LDA, 
			adj_csr.d_nnzPerVector, 
			adj_csr.d_values, adj_csr.d_rowIndices, adj_csr.d_colIndices); 
	device2host(&adj_csr, adj_csr.nnz, g.size);	
	if(DEBUG){
		print_csr(
    			g.size,
    			adj_csr.nnz,
    			adj_csr,
    			"Adjacency matrix");
	}

	allocate_csr(&degree_csr, degree_csr.nnz, g.size);
	hipsparseDdense2csr(cusparse_handle, 
			g.size, g.size, 
			mat_descr, 
			degree_device_dense,
			LDA, 
			degree_csr.d_nnzPerVector, 
			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices); 
	device2host(&degree_csr, degree_csr.nnz, g.size);	

	if(DEBUG){
		print_csr(
    			g.size,
    			degree_csr.nnz,
    			degree_csr,
    			"Degree matrix");
	}

	log("Completed conversion of data from dense to sparse");

	/* Section 2: Compute X = D^{-1/2} * A * D^{-1/2} */
	/* Procedure
	   1. Compute D' = D^{-1/2}
	   2. Compute X' = D' * A
	   3. Compute X = X' * D'
	*/
	
	/* Step 1: Compute D' = D^{-1/2} */
	log("Computing normalized D");
	compute_d<<<grids, threads>>>(degree_csr.d_values, degree_csr.nnz);
	
	log("Computed normalized D");
	if(DEBUG){
		device2host(&degree_csr, degree_csr.nnz, g.size);
		print_csr(
			g.size,
			degree_csr.nnz,
			degree_csr,
			"Normalized Degree Matrix");
	}
			

	/* Step 2: Compute X' = D' * A */
	csr X_temp;
	multiply_csr(&adj_csr, &degree_csr, &X_temp, g.size, g.size, g.size, cusparse_handle, mat_descr);
	
	if(DEBUG){
		device2host(&X_temp, X_temp.nnz, g.size);
		print_csr(g.size, X_temp.nnz, X_temp, "X' = D' * A");
	}
	


//	
//	// Compute X = X_tempD^{-1/2} second
//	hipsparseMatDescr_t X_descr;
//	hipsparseCreateMatDescr(&X_descr);
//	hipsparseSetMatType(X_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//	hipsparseSetMatIndexBase(X_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//	csr X_csr;
//       	X_csr.nnz = 0;
//	nnzTotalDevHostPtr = &X_csr.nnz;
//
//	hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
//	hipMalloc(&X_csr.d_rowIndices, (g.size + 1) * sizeof(int));
//	status = hipsparseXcsrgemmNnz(cusparse_handle, 
//				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//				g.size, g.size, g.size,
//				X_temp_descr, X_temp_csr.nnz,
//				X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices, 
//				degree_descr, degree_csr.nnz,
//				degree_csr.d_rowIndices, degree_csr.d_colIndices,
//				X_descr, X_csr.d_rowIndices,
//				nnzTotalDevHostPtr
//				);
//	if (status != HIPSPARSE_STATUS_SUCCESS) {
//		std::cout << "nnz calculation failed" << std::endl;
//		std::cout << "status = " << status << std::endl;
//		exit(0);
//	}		
//	//std::cout<<"Value of NNZ: "<<*nnzTotalDevHostPtr<<std::endl;
//
//	if(NULL != nnzTotalDevHostPtr){
//		X_csr.nnz = *nnzTotalDevHostPtr;
//	}else{
//		hipMemcpy(&X_csr.nnz, X_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
//		hipMemcpy(&baseX, X_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
//		X_csr.nnz -= baseX;
//	}
//
//	hipMalloc(&X_csr.d_colIndices, X_csr.nnz * sizeof(int));
//	hipMalloc(&X_csr.d_rowIndices, (g.size + 1) * sizeof(int));
//	hipMalloc(&X_csr.d_values, X_csr.nnz * sizeof(double));
//
//	X_csr.h_colIndices = (int *) malloc(X_csr.nnz * sizeof(int));
//	X_csr.h_rowIndices = (int *) malloc((g.size +1) * sizeof(int));
//	X_csr.h_values = (double *) malloc(X_csr.nnz * sizeof(double));
//
//	hipsparseDcsrgemm(cusparse_handle, 
//			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//			g.size, g.size, g.size,
//			X_temp_descr, X_temp_csr.nnz,
//			X_temp_csr.d_values, X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices,
//			degree_descr, degree_csr.nnz,
//			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices,
//			X_descr, 
//			X_csr.d_values, X_csr.d_rowIndices, X_csr.d_colIndices);
//	hipDeviceSynchronize();
//
//	cuda_status = hipMemcpy(X_csr.h_values, X_csr.d_values, X_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//	cuda_status = hipMemcpy(X_csr.h_rowIndices, X_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//	cuda_status = hipMemcpy(X_csr.h_colIndices, X_csr.d_colIndices, X_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//
//	//std::cout<<cuda_status;
//
//	log("Print X");
////	for(int i=0;i<X_csr.nnz;i++) std::cout<<X_csr.h_values[i]<< " "; std::cout<<"\n";
////	for(int i=0;i<X_csr.nnz;i++) std::cout<<X_csr.h_colIndices[i]<< " "; std::cout<<"\n";
////	for(int i=0;i<g.size + 1;i++) std::cout<<X_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
//
//
//	 /* Compute S = sum(X^{0}....X^{window_size}) */
//
//	csr S_csr, temp_csr, temp1_csr, S_temp_csr;
//	S_csr.nnz = X_csr.nnz;
//	temp_csr.nnz = X_csr.nnz;
//
//	hipMalloc(&S_csr.d_values, S_csr.nnz * sizeof(double));
//	hipMalloc(&S_csr.d_colIndices, S_csr.nnz * sizeof(int));
//	hipMalloc(&S_csr.d_rowIndices, (g.size + 1) * sizeof(int));
//
//	hipMalloc(&temp_csr.d_values, temp_csr.nnz * sizeof(double));
//	hipMalloc(&temp_csr.d_colIndices, temp_csr.nnz * sizeof(int));
//	hipMalloc(&temp_csr.d_rowIndices, (g.size + 1) * sizeof(int));
//
//	log("Copying X to S");
//	hipMemcpy(S_csr.d_values, X_csr.d_values, S_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
//	hipMemcpy(S_csr.d_colIndices, X_csr.d_colIndices, S_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);
//	hipMemcpy(S_csr.d_rowIndices, X_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);
//
//	log("Copying S to host");
//	S_csr.h_values = (double *) malloc(S_csr.nnz * sizeof(double));
//	S_csr.h_colIndices = (int *) malloc(S_csr.nnz * sizeof(int));
//	S_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(int));
//
//	hipMemcpy(S_csr.h_values, S_csr.d_values, S_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//	hipMemcpy(S_csr.h_colIndices, S_csr.d_colIndices, S_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//	hipMemcpy(S_csr.h_rowIndices, S_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//
//	//for(int i=0;i<S_csr.nnz;i++) std::cout<<S_csr.h_values[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<S_csr.nnz;i++) std::cout<<S_csr.h_colIndices[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<g.size + 1;i++) std::cout<<S_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
//
//	log("Copying X to temp");
//	hipMemcpy(temp_csr.d_values, X_csr.d_values, temp_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
//	hipMemcpy(temp_csr.d_colIndices, X_csr.d_colIndices, temp_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);
//	hipMemcpy(temp_csr.d_rowIndices, X_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);
//
//	log("Copying S to host");
//	temp_csr.h_values = (double *) malloc(temp_csr.nnz * sizeof(double));
//	temp_csr.h_colIndices = (int *) malloc(temp_csr.nnz * sizeof(int));
//	temp_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(int));
//
//	hipMemcpy(temp_csr.h_values, temp_csr.d_values, temp_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//	hipMemcpy(temp_csr.h_colIndices, temp_csr.d_colIndices, temp_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//	hipMemcpy(temp_csr.h_rowIndices, temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//
//	//for(int i=0;i<temp_csr.nnz;i++) std::cout<<temp_csr.h_values[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<temp_csr.nnz;i++) std::cout<<temp_csr.h_colIndices[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<g.size + 1;i++) std::cout<<temp_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
//
//
//	hipsparseMatDescr_t S_descr;
//	hipsparseCreateMatDescr(&S_descr);
//	hipsparseSetMatType(S_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//	hipsparseSetMatIndexBase(S_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//	hipsparseMatDescr_t S_temp_descr;
//	hipsparseCreateMatDescr(&S_temp_descr);
//	hipsparseSetMatType(S_temp_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//	hipsparseSetMatIndexBase(S_temp_descr, HIPSPARSE_INDEX_BASE_ZERO);
//	
//	hipsparseMatDescr_t temp_descr;
//	hipsparseCreateMatDescr(&temp_descr);
//	hipsparseSetMatType(temp_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//	hipsparseSetMatIndexBase(temp_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//	hipsparseMatDescr_t temp1_descr;
//	hipsparseCreateMatDescr(&temp1_descr);
//	hipsparseSetMatType(temp1_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//	hipsparseSetMatIndexBase(temp1_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//	double alf = 1.00;
//	double beta = 1.00;
//
//	for(int i=2;i<=window_size;i++){
//		std::cout<<"Computing X^"<<i<<std::endl;
//		temp1_csr.nnz = 0;
//		temp1_csr.d_rowIndices = NULL;
//		temp1_csr.d_colIndices = NULL;
//		temp1_csr.d_values = NULL;
//		
//		temp1_csr.h_rowIndices = NULL;
//		temp1_csr.h_colIndices = NULL;
//		temp1_csr.h_values = NULL;
//		
//		S_temp_csr.nnz = 0;
//		S_temp_csr.d_rowIndices = NULL;
//		S_temp_csr.d_colIndices = NULL;
//		S_temp_csr.d_values = NULL;
//		
//		S_temp_csr.h_rowIndices = NULL;
//		S_temp_csr.h_colIndices = NULL;
//		S_temp_csr.h_values = NULL;
//
//		nnzTotalDevHostPtr = &temp1_csr.nnz;
//
//		hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
//		hipMalloc(&temp1_csr.d_rowIndices, (g.size + 1) * sizeof(double));
//		status = hipsparseXcsrgemmNnz(cusparse_handle, 
//					HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//					g.size, g.size, g.size,
//					temp_descr, temp_csr.nnz,
//					temp_csr.d_rowIndices, temp_csr.d_colIndices,
//					X_descr, X_csr.nnz,
//					X_csr.d_rowIndices, X_csr.d_colIndices,
//					temp1_descr, temp1_csr.d_rowIndices,
//					nnzTotalDevHostPtr);
//
//		if(status != 0){
//			std::cout<<"Error encountered"<<std::endl;
//			std::cout<<"Status: "<<status<<std::endl;
//			exit(0);	
//
//		}
//
//
//		if(NULL!=nnzTotalDevHostPtr) temp1_csr.nnz = *nnzTotalDevHostPtr;
//		else{
//			hipMemcpy(&temp1_csr.nnz, temp1_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
//			hipMemcpy(&baseX, temp1_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
//			temp1_csr.nnz -= baseX;
//		
//		}
//
//		//std::cout<<"NNZ ="<<temp1_csr.nnz<<std::endl;
//
//
//		hipMalloc(&temp1_csr.d_values, temp1_csr.nnz * sizeof(double));
//		hipMalloc(&temp1_csr.d_colIndices, temp1_csr.nnz * sizeof(int));
//
//		hipsparseDcsrgemm(cusparse_handle,
//				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//				g.size, g.size, g.size,
//				temp_descr, temp_csr.nnz,
//				temp_csr.d_values, temp_csr.d_rowIndices, temp_csr.d_colIndices,
//				X_descr, X_csr.nnz,
//				X_csr.d_values, X_csr.d_rowIndices, X_csr.d_colIndices,
//				temp1_descr, 
//				temp1_csr.d_values, temp1_csr.d_rowIndices, temp1_csr.d_colIndices);
//	
//		log("Printing intermediate result");					
//		temp1_csr.h_values = (double *) malloc(temp1_csr.nnz * sizeof(double));
//		temp1_csr.h_colIndices = (int *) malloc(temp1_csr.nnz * sizeof(int));
//		temp1_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(int));
//
//		hipMemcpy(temp1_csr.h_values, temp1_csr.d_values, temp1_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//		hipMemcpy(temp1_csr.h_colIndices, temp1_csr.d_colIndices, temp1_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//		hipMemcpy(temp1_csr.h_rowIndices, temp1_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//
//	//	for(int i=0;i<temp1_csr.nnz;i++) std::cout<<temp1_csr.h_values[i]<< " "; std::cout<<"\n";
//	//	for(int i=0;i<temp1_csr.nnz;i++) std::cout<<temp1_csr.h_colIndices[i]<< " "; std::cout<<"\n";
//	//	for(int i=0;i<g.size + 1;i++) std::cout<<temp1_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
//
//		hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
//
//		S_temp_csr.nnz = 0;
//		nnzTotalDevHostPtr = &S_temp_csr.nnz;
//		hipMalloc(&S_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int));
//		
//		status = hipsparseXcsrgeamNnz(cusparse_handle,
//					g.size, g.size,
//					temp1_descr, temp1_csr.nnz, temp1_csr.d_rowIndices, temp1_csr.d_colIndices,
//					S_descr,S_csr.nnz, S_csr.d_rowIndices, S_csr.d_colIndices,
//					S_temp_descr, S_temp_csr.d_rowIndices, nnzTotalDevHostPtr);	
//
//		if(status != 0){
//			std::cout<<"Error encountered"<<std::endl;
//			std::cout<<"Status: "<<status<<std::endl;
//			exit(0);	
//
//		}
//		if(NULL!=nnzTotalDevHostPtr) S_temp_csr.nnz = *nnzTotalDevHostPtr;
//		else{
//			hipMemcpy(&S_temp_csr.nnz, S_temp_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
//			hipMemcpy(&baseX, S_temp_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
//			S_temp_csr.nnz -= baseX;
//		}
//
//		//std::cout<<"SUM-NNZ"<<S_temp_csr.nnz<<std::endl;
//
//		hipMalloc(&S_temp_csr.d_values, sizeof(double) * S_temp_csr.nnz);
//		hipMalloc(&S_temp_csr.d_colIndices, sizeof(int) * S_temp_csr.nnz);
//
//		hipsparseDcsrgeam(cusparse_handle, g.size, g.size,
//				&alf,
//				temp1_descr, temp1_csr.nnz,
//				temp1_csr.d_values, temp1_csr.d_rowIndices, temp1_csr.d_colIndices,
//				&beta,
//				S_descr, S_csr.nnz,
//				S_csr.d_values, S_csr.d_rowIndices, S_csr.d_colIndices,
//				S_temp_descr,
//				S_temp_csr.d_values, S_temp_csr.d_rowIndices, S_temp_csr.d_colIndices);
//
//		log("Printing intermediate sum");
//		S_temp_csr.h_values = (double *) malloc(sizeof(double) * S_temp_csr.nnz);
//		S_temp_csr.h_rowIndices = (int *) malloc(sizeof(int) * (g.size + 1));
//		S_temp_csr.h_colIndices = (int *) malloc(sizeof(int) * S_temp_csr.nnz);
//
//		hipMemcpy(S_temp_csr.h_values, S_temp_csr.d_values, S_temp_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//		hipMemcpy(S_temp_csr.h_rowIndices, S_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//		hipMemcpy(S_temp_csr.h_colIndices, S_temp_csr.d_colIndices, S_temp_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//
//		//for(int i=0;i<S_temp_csr.nnz;i++) std::cout<<S_temp_csr.h_values[i]<< " "; std::cout<<"\n";
//		//for(int i=0;i<S_temp_csr.nnz;i++) std::cout<<S_temp_csr.h_colIndices[i]<< " "; std::cout<<"\n";
//		//for(int i=0;i<g.size + 1;i++) std::cout<<S_temp_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
//
//		S_csr.nnz = S_temp_csr.nnz;
//		hipFree(S_csr.d_values);
//		hipFree(S_csr.d_rowIndices);
//		//hipFree(S_csr.d_colIndices);
//		
//		hipMalloc(&S_csr.d_values, S_csr.nnz * sizeof(double)); 
//		hipMalloc(&S_csr.d_rowIndices, (g.size + 1) * sizeof(int)); 
//		hipMalloc(&S_csr.d_colIndices, S_csr.nnz * sizeof(int)); 
//
//		hipMemcpy(S_csr.d_values, S_temp_csr.d_values, S_temp_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
//		hipMemcpy(S_csr.d_rowIndices, S_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);
//		hipMemcpy(S_csr.d_colIndices, S_temp_csr.d_colIndices, S_temp_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);
//
//
//		temp_csr.nnz = temp1_csr.nnz;
//		hipFree(temp_csr.d_values);
//		hipFree(temp_csr.d_rowIndices);
//		//hipFree(temp_csr.d_colIndices);
//
//		hipMalloc(&temp_csr.d_values, temp_csr.nnz * sizeof(double)); 
//		hipMalloc(&temp_csr.d_rowIndices, (g.size + 1) * sizeof(int)); 
//		hipMalloc(&temp_csr.d_colIndices, temp_csr.nnz * sizeof(int)); 
//
//		hipMemcpy(temp_csr.d_values, temp1_csr.d_values, temp_csr.nnz * sizeof(double), hipMemcpyDeviceToDevice);
//		hipMemcpy(temp_csr.d_rowIndices, temp1_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToDevice);
//		hipMemcpy(temp_csr.d_colIndices, temp1_csr.d_colIndices, temp_csr.nnz * sizeof(int), hipMemcpyDeviceToDevice);
//			
//	}
//
//        /* Compute S = S * (vol / (window_size * b)) */
//
//	transform_s<<<grids, threads>>>(S_csr.d_values, g.volume, window_size, b, S_csr.nnz);
//
//	S_csr.h_values = (double *)malloc(sizeof(double) * S_csr.nnz);
//
//	hipMemcpy(S_csr.h_values, S_csr.d_values, S_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//
//	log("Transformed S values");
//	//for(int i=0;i<S_csr.nnz;i++) std::cout<<S_csr.h_values[i]<< " "; std::cout<<"\n";
//
//	log("Computing M");
//
//
//        /* Compute M = D^{-1/2} * S * D^{-1/2} */
//
//	// Compute X_temp = D^{-1/2} * S first	
//       	X_temp_csr.nnz = 0;
//	nnzTotalDevHostPtr = &X_temp_csr.nnz;
//
//	hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
//	hipMalloc(&X_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int));
//	status = hipsparseXcsrgemmNnz(cusparse_handle, 
//				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//				g.size, g.size, g.size,
//				degree_descr, degree_csr.nnz,
//				degree_csr.d_rowIndices, degree_csr.d_colIndices, 
//				S_descr, S_csr.nnz,
//				S_csr.d_rowIndices, S_csr.d_colIndices,
//				X_temp_descr, X_temp_csr.d_rowIndices,
//				nnzTotalDevHostPtr
//				);
//	if (status != HIPSPARSE_STATUS_SUCCESS) {
//		std::cout << "nnz calculation failed" << std::endl;
//		std::cout << "status = " << status << std::endl;
//		exit(0);
//	}		
//	//std::cout<<"Value of NNZ: "<<*nnzTotalDevHostPtr<<std::endl;
//
//	if(NULL != nnzTotalDevHostPtr){
//		X_temp_csr.nnz = *nnzTotalDevHostPtr;
//	}else{
//		hipMemcpy(&X_temp_csr.nnz, X_temp_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
//		hipMemcpy(&baseX, X_temp_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
//		X_temp_csr.nnz -= baseX;
//	}
//
//	hipMalloc(&X_temp_csr.d_colIndices, X_temp_csr.nnz * sizeof(int));
//	hipMalloc(&X_temp_csr.d_values, X_temp_csr.nnz * sizeof(double));
//
//	X_temp_csr.h_colIndices = (int *) malloc(X_temp_csr.nnz * sizeof(double));
//	X_temp_csr.h_rowIndices = (int *) malloc((g.size + 1) * sizeof(double));
//	X_temp_csr.h_values = (double *) malloc(X_temp_csr.nnz * sizeof(double));
//
//	hipsparseDcsrgemm(cusparse_handle, 
//			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//			g.size, g.size, g.size,
//			degree_descr, degree_csr.nnz,
//			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices,
//			S_descr, S_csr.nnz,
//			S_csr.d_values, S_csr.d_rowIndices, S_csr.d_colIndices,
//			X_temp_descr, 
//			X_temp_csr.d_values, X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices);
//	hipDeviceSynchronize();
//
//	cuda_status = hipMemcpy(X_temp_csr.h_values, X_temp_csr.d_values, X_temp_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//	cuda_status = hipMemcpy(X_temp_csr.h_rowIndices, X_temp_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//	cuda_status = hipMemcpy(X_temp_csr.h_colIndices, X_temp_csr.d_colIndices, X_temp_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//
//	log("Print X_temp");
//	//for(int i=0;i<X_temp_csr.nnz;i++) std::cout<<X_temp_csr.h_values[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<X_temp_csr.nnz;i++) std::cout<<X_temp_csr.h_colIndices[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<g.size + 1;i++) std::cout<<X_temp_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
//	
//	// Compute M = X_temp * D^{-1/2} second
//	hipsparseMatDescr_t M_descr;
//	hipsparseCreateMatDescr(&M_descr);
//	hipsparseSetMatType(M_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//	hipsparseSetMatIndexBase(M_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//	csr M_csr;
//       	M_csr.nnz = 0;
//	nnzTotalDevHostPtr = &M_csr.nnz;
//
//	hipsparseSetPointerMode(cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
//	hipMalloc(&M_csr.d_rowIndices, (g.size + 1) * sizeof(int));
//	status = hipsparseXcsrgemmNnz(cusparse_handle, 
//				HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//				g.size, g.size, g.size,
//				X_temp_descr, X_temp_csr.nnz,
//				X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices, 
//				degree_descr, degree_csr.nnz,
//				degree_csr.d_rowIndices, degree_csr.d_colIndices,
//				M_descr, M_csr.d_rowIndices,
//				nnzTotalDevHostPtr
//				);
//	if (status != HIPSPARSE_STATUS_SUCCESS) {
//		std::cout << "nnz calculation failed" << std::endl;
//		std::cout << "status = " << status << std::endl;
//		exit(0);
//	}		
//	//std::cout<<"Value of NNZ: "<<*nnzTotalDevHostPtr<<std::endl;
//
//	if(NULL != nnzTotalDevHostPtr){
//		M_csr.nnz = *nnzTotalDevHostPtr;
//	}else{
//		hipMemcpy(&M_csr.nnz, M_csr.d_rowIndices + g.size, sizeof(int), hipMemcpyDeviceToHost);
//		hipMemcpy(&baseX, M_csr.d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
//		M_csr.nnz -= baseX;
//	}
//
//	hipMalloc(&M_csr.d_colIndices, M_csr.nnz * sizeof(int));
//	hipMalloc(&M_csr.h_rowIndices, (g.size + 1) * sizeof(int));
//	hipMalloc(&M_csr.d_values, M_csr.nnz * sizeof(double));
//
//	M_csr.h_colIndices = (int *) malloc(M_csr.nnz * sizeof(double));
//	M_csr.h_rowIndices = (int *) malloc(M_csr.nnz * sizeof(double));
//	M_csr.h_values = (double *) malloc(M_csr.nnz * sizeof(double));
//
//	hipsparseDcsrgemm(cusparse_handle, 
//			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//			g.size, g.size, g.size,
//			X_temp_descr, X_temp_csr.nnz,
//			X_temp_csr.d_values, X_temp_csr.d_rowIndices, X_temp_csr.d_colIndices,
//			degree_descr, degree_csr.nnz,
//			degree_csr.d_values, degree_csr.d_rowIndices, degree_csr.d_colIndices,
//			M_descr, 
//			M_csr.d_values, M_csr.d_rowIndices, M_csr.d_colIndices);
//	hipDeviceSynchronize();
//
//	cuda_status = hipMemcpy(M_csr.h_values, M_csr.d_values, M_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//	cuda_status = hipMemcpy(M_csr.h_rowIndices, M_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//	cuda_status = hipMemcpy(M_csr.h_colIndices, M_csr.d_colIndices, M_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//
//	//std::cout<<cuda_status;
//
//	log("Print M");
//	//for(int i=0;i<M_csr.nnz;i++) std::cout<<M_csr.h_values[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<M_csr.nnz;i++) std::cout<<M_csr.h_colIndices[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<g.size + 1;i++) std::cout<<M_csr.h_rowIndices[i]<< " "; std::cout<<"\n";
//
//	log("Transforming M");
//
//	prune_m<<<grids,threads>>>(M_csr.d_values, M_csr.nnz);
//	hipMemcpy(M_csr.h_values, M_csr.d_values, M_csr.nnz * sizeof(double), hipMemcpyDeviceToHost);
//	hipMemcpy(M_csr.h_rowIndices, M_csr.d_rowIndices, (g.size + 1) * sizeof(int), hipMemcpyDeviceToHost);
//	hipMemcpy(M_csr.h_colIndices, M_csr.d_colIndices, M_csr.nnz * sizeof(int), hipMemcpyDeviceToHost);
//	//for(int i=0;i<M_csr.nnz;i++) std::cout<<M_csr.h_values[i]<< " "; std::cout<<"\n";
//	hipDeviceSynchronize();
//
//
//
//	double threshold = 1.00;
//	log("Setting threshold");
//	//std::cout<<"Threshold:"<<threshold;
//	csr filtered_M;
//
//	hipMalloc(&filtered_M.d_rowIndices, sizeof(int) * (g.size + 1));
//	size_t lworkInBytes = 0;
//	char *d_work=NULL;
//	hipsparseDpruneCsr2csr_bufferSizeExt(cusparse_handle,
//        					g.size,g.size,
//        					M_csr.nnz,M_descr,
//        					M_csr.d_values,M_csr.d_rowIndices,M_csr.d_colIndices,
//        					&threshold,
//        					M_descr,
//        					filtered_M.d_values, filtered_M.d_rowIndices, filtered_M.d_colIndices,
//        					&lworkInBytes);	
//
//    	//printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);
//	hipMalloc((void**)&d_work, lworkInBytes);
//
//	hipsparseDpruneCsr2csrNnz(cusparse_handle,
//					g.size,g.size,
//        				M_csr.nnz,M_descr,
//  					M_csr.d_values,M_csr.d_rowIndices,M_csr.d_colIndices,
//        				&threshold,
//        				M_descr,
//        				filtered_M.d_rowIndices, &filtered_M.nnz, /* host */
//        				d_work);
//
//	//printf("nnzC = %d\n", filtered_M.nnz);
//    	if (0 == filtered_M.nnz ){
//        	printf("C is empty \n");
//        	return 0;
//    	}
//
//
//	hipMalloc(&filtered_M.d_colIndices, sizeof(int) * filtered_M.nnz);
//	hipMalloc(&filtered_M.d_values, sizeof(double) * filtered_M.nnz);
//
//	hipsparseDpruneCsr2csr(cusparse_handle,
//   				g.size,g.size,
//        			M_csr.nnz, M_descr,
//        			M_csr.d_values, M_csr.d_rowIndices, M_csr.d_colIndices,
//        			&threshold,
//        			M_descr,
//        			filtered_M.d_values,filtered_M.d_rowIndices, filtered_M.d_colIndices,
//        			d_work);
//
//
//	log("Printing pruned M");
//	filtered_M.h_values = (double *) malloc(sizeof(double) * filtered_M.nnz);
//	filtered_M.h_colIndices = (int *) malloc(sizeof(int) * filtered_M.nnz);
//	filtered_M.h_rowIndices = (int *) malloc(sizeof(int) * (g.size + 1));
//
//	hipMemcpy(filtered_M.h_values, filtered_M.d_values, sizeof(double) * filtered_M.nnz, hipMemcpyDeviceToHost);
//	hipMemcpy(filtered_M.h_colIndices, filtered_M.d_colIndices, sizeof(int) * filtered_M.nnz, hipMemcpyDeviceToHost);
//	hipMemcpy(filtered_M.h_rowIndices, filtered_M.d_rowIndices, sizeof(int) * (g.size + 1), hipMemcpyDeviceToHost);
//	
//	//for(int i=0;i<filtered_M.nnz;i++) std::cout<<filtered_M.h_values[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<filtered_M.nnz;i++) std::cout<<filtered_M.h_colIndices[i]<< " "; std::cout<<"\n";
//	//for(int i=0;i<g.size + 1;i++) std::cout<<filtered_M.h_rowIndices[i]<< " "; std::cout<<"\n";
//	
//	log("Printing log of M");
//	transform_m<<<grids,threads>>>(filtered_M.d_values, filtered_M.nnz);
//
//	hipMemcpy(filtered_M.h_values, filtered_M.d_values, sizeof(double) * filtered_M.nnz, hipMemcpyDeviceToHost);
//	hipMemcpy(filtered_M.h_colIndices, filtered_M.d_colIndices, sizeof(int) * filtered_M.nnz, hipMemcpyDeviceToHost);
//	hipMemcpy(filtered_M.h_rowIndices, filtered_M.d_rowIndices, sizeof(int) * (g.size + 1), hipMemcpyDeviceToHost);
//	//for(int i=0;i<filtered_M.nnz;i++) std::cout<<filtered_M.h_values[i]<< " "; std::cout<<"\n";
//
//	char whichS = 'L';
//	char whichV = 'L';
//
//	MKL_INT pm[128];
//	mkl_sparse_ee_init(pm);
//	//pm[1] = 100;
//	//pm[2] = 2;
//	//pm[4] = 60;
//
//	MKL_INT mkl_rows = g.size;
//	MKL_INT mkl_cols = g.size;
//
//
//	MKL_INT rows_start[mkl_rows];
//	MKL_INT rows_end[mkl_rows];
//
//	for(int i=0;i<mkl_rows;i++){
//		rows_start[i] = filtered_M.h_rowIndices[i];
//		rows_end[i] = filtered_M.h_colIndices[i];
//	}
//
//	
//	MKL_INT mkl_col_idx[filtered_M.nnz];
//	for(int i=0;i<filtered_M.nnz;i++)
//		mkl_col_idx[i] = filtered_M.h_colIndices[i];
//
//
//	sparse_matrix_t M_mkl;
//	sparse_index_base_t indexing = SPARSE_INDEX_BASE_ZERO;
//
//	mkl_sparse_d_create_csr(&M_mkl, indexing,
//					mkl_rows, mkl_cols,
//					rows_start, rows_end,
//					mkl_col_idx, filtered_M.h_values);
//
//	log("Created MKL sparse");
//
//	matrix_descr mkl_descrM;
//	mkl_descrM.type = SPARSE_MATRIX_TYPE_GENERAL;	
//	mkl_descrM.mode = SPARSE_FILL_MODE_UPPER;
//	mkl_descrM.diag = SPARSE_DIAG_NON_UNIT;
//
//	MKL_INT k0 = dimension;
//	MKL_INT k;
//
//	double *E_mkl, *K_L_mkl, *K_R_mkl, *res_mkl;
//
//	E_mkl = (double *)mkl_malloc(k0 * sizeof(double), 128);
//	K_L_mkl = (double *)mkl_malloc( k0*mkl_rows*sizeof( double), 128 );
//        K_R_mkl = (double *)mkl_malloc( k0*mkl_cols*sizeof( double), 128 );
//        res_mkl = (double *)mkl_malloc( k0*sizeof( double), 128 );
//
//	memset(E_mkl, 0 , k0);
//	memset(K_L_mkl, 0 , k0);
//	memset(K_R_mkl, 0 , k0);
//	memset(res_mkl, 0 , k0);
//
//	int mkl_status = 0;
//
//	log("Computing SVD via MKL");
//	mkl_status = mkl_sparse_d_svd(&whichS, &whichV, pm,
//			M_mkl, mkl_descrM,
//			k0, &k,
//			E_mkl,
//			K_L_mkl,
//			K_R_mkl,
//			res_mkl);
//	log("Computed SVD via MKL");
//
//	std::cout<<"Number of eigenvalues found: "<<k<<std::endl;
//	for(int i=0;i<k0;i++){ std::cout<<E_mkl[i]<<" ";} std::cout<<"\n";
//
//
}
