#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<iostream>
#include<time.h>
#include<chrono>
#include<algorithm>
#include<numeric>
#include<math.h>

#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include <hipsparse.h> 

#include "../utils/graph.h"
#include "../utils/io.h"

#include<mkl.h>
#include<mkl_solvers_ee.h>
#include<mkl_spblas.h>

#define DEBUG true
#define VERBOSE false


void print_csr(
    int m,
    int nnz,
    csr mat_csr,
    const char* name)
{
    printf("matrix %s is %d-by-%d, nnz=%d\n", name, m, m, nnz);
    std::cout<<"Values: "; for(int i=0;i<nnz;i++) std::cout<<mat_csr.h_values[i]<<" "; std::cout<<'\n';
    std::cout<<"Cols: "; for(int i=0;i<nnz;i++) std::cout<<mat_csr.h_colIndices[i]<<" "; std::cout<<'\n';
    std::cout<<"Rows: "; for(int i=0;i<m+1;i++) std::cout<<mat_csr.h_rowIndices[i]<<" "; std::cout<<'\n';
}
__global__ void preprocess_laplacian(double* adj, double *degree, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= size) return;

	// Remove self loops
	// If deg(v) = 0 -> deg(0) = 1
		
	if(degree[id] == 0){
			degree[id] = 1.00;
			adj[id*size + id] = 1.00;	
	}else{
			adj[id * size + id] = 0.0;
	}	
}
__global__ void compute_d(double* deg, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= size) return;

	//id = id * size + id;
	
	/* Make assumption here that graph is  	*/
	/* connected and every node has degree 	*/
        /* atleast 1. 		       		*/
	
	if(deg[id] == -1)
		deg[id] = 0;
	else	
		deg[id] = 1 / sqrt(deg[id]); 
}

__global__ void compute_s(float* S, float* X, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] += X[id]; 
}

__global__ void transform_si(double* S, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = sqrt(S[id]); 
}

__global__ void transform_s(double* S, float val, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;

	double mem = S[id];
	
	S[id] = mem * val; 
}

__global__ void prune_m(double* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;

	if(M[id] <= 1)
		M[id] = 0;
	else
		M[id] = log(M[id]);	
}


__global__ void transform_m(double* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	M[id] =log(M[id]);
}

__global__ void sqrt_si(float* S, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = sqrt((float) S[id]);
}

void print_matrix(double* S, int size){
	std::cout<<std::endl<<std::endl;
	for(int i=0;i<size;i++){
		for(int j=0;j<size;j++){
			std::cout<<S[i*size + j]<<" ";
		}
		std::cout<<std::endl;
	}

} 
void allocate_csr_row(csr *csr_mat, int num_rows){
	csr_mat->h_rowIndices = (int *) malloc((num_rows+1) * sizeof(int));
	hipMalloc(&csr_mat->d_rowIndices, (num_rows+1) * sizeof(int));

}
void allocate_csr_col_val(csr *csr_mat, int nnz){
	csr_mat->h_values = (double *) malloc(nnz * sizeof(double));
	csr_mat->h_colIndices = (int *) malloc(nnz * sizeof(int));

	hipMalloc(&csr_mat->d_values, nnz*sizeof(double));
	hipMalloc(&csr_mat->d_colIndices, nnz * sizeof(int));

}

void allocate_csr(csr *csr_mat, int nnz, int num_rows){
	csr_mat->h_values = (double *) malloc(nnz * sizeof(double));
	csr_mat->h_colIndices = (int *) malloc(nnz * sizeof(int));
	csr_mat->h_rowIndices = (int *) malloc((num_rows+1) * sizeof(int));

	hipMalloc(&csr_mat->d_values, nnz*sizeof(double));
	hipMalloc(&csr_mat->d_colIndices, nnz * sizeof(int));
	hipMalloc(&csr_mat->d_rowIndices, (num_rows+1) * sizeof(int));

}

void free_csr(csr *A){
	free(A->h_values);
	free(A->h_rowIndices);
	free(A->h_colIndices);

	hipFree(A->d_values);
	hipFree(A->d_rowIndices);
	hipFree(A->d_colIndices);
}

void copy_csr(csr *from_mat, csr *to_mat, int num_rows){
	to_mat->nnz = from_mat->nnz;

	/* Copy host variables */
	memcpy(to_mat->h_values, from_mat->h_values, to_mat->nnz * sizeof(double));
	memcpy(to_mat->h_colIndices, from_mat->h_colIndices, to_mat->nnz * sizeof(int));
	memcpy(to_mat->h_rowIndices, from_mat->h_rowIndices, (num_rows + 1) * sizeof(int));


	/* Copy device variables */
	hipMemcpy(to_mat->d_values, from_mat->d_values, to_mat->nnz * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(to_mat->d_colIndices, from_mat->d_colIndices, to_mat->nnz * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(to_mat->d_rowIndices, from_mat->d_rowIndices, (num_rows + 1) * sizeof(int), hipMemcpyDeviceToDevice);

}

void device2host(csr *csr_mat, int nnz, int num_rows){
	hipMemcpy(csr_mat->h_values, csr_mat->d_values, 
			nnz * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(csr_mat->h_colIndices, csr_mat->d_colIndices, 
			nnz * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(csr_mat->h_rowIndices, csr_mat->d_rowIndices, 
			(num_rows + 1) * sizeof(int), hipMemcpyDeviceToHost);
}

void host2device(csr *csr_mat, int nnz, int num_rows){
	hipMemcpy(csr_mat->d_values, csr_mat->h_values, 
			nnz * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(csr_mat->d_colIndices, csr_mat->h_colIndices, 
			nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(csr_mat->d_rowIndices, csr_mat->h_rowIndices, 
			(num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
}

void add_csr(csr *A, csr *B, csr *C, int m, int n, hipsparseHandle_t context,hipsparseMatDescr_t descr){

	hipsparseStatus_t status;

	int base;
	int *nnzTotalDevHostPtr;
	C->nnz = 0;
	nnzTotalDevHostPtr = &C->nnz;

	double alf = 1.0;
	double bet = 1.0;

	hipsparseSetPointerMode(context, HIPSPARSE_POINTER_MODE_HOST);
	allocate_csr_row(C, m);
		
	status = hipsparseXcsrgeamNnz(context,
				m, n,
				descr, A->nnz, A->d_rowIndices, A->d_colIndices,
				descr, B->nnz, B->d_rowIndices, B->d_colIndices,
				descr, C->d_rowIndices, nnzTotalDevHostPtr);	

	if(status != HIPSPARSE_STATUS_SUCCESS){
		std::cout<<"Error encountered"<<std::endl;
		std::cout<<"Status: "<<status<<std::endl;
		exit(0);	
	}
	
	if(NULL != nnzTotalDevHostPtr){
		C->nnz = *nnzTotalDevHostPtr;
	}else{
		hipMemcpy(&C->nnz, C->d_rowIndices + m, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&base, C->d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
		C->nnz -= base;
	}

	allocate_csr_col_val(C, C->nnz);	

	hipsparseDcsrgeam(context, m, n,
			&alf,
			descr, A->nnz,
			A->d_values, A->d_rowIndices, A->d_colIndices,
			&bet,
			descr, B->nnz,
			B->d_values, B->d_rowIndices, B->d_colIndices,
			descr,
			C->d_values, C->d_rowIndices, C->d_colIndices);

}

void multiply_csr(csr *A, csr *B, csr *C, int m, int n, int k, hipsparseHandle_t context,hipsparseMatDescr_t descr ){

	hipsparseStatus_t status;

	int base;
	int *nnzTotalDevHostPtr;
	C->nnz = 0;
	nnzTotalDevHostPtr = &C->nnz;

	hipsparseSetPointerMode(context, HIPSPARSE_POINTER_MODE_HOST);

	allocate_csr_row(C, m);
	//hipMalloc(&C->d_rowIndices, (m+1) * sizeof(int));
	
	status = hipsparseXcsrgemmNnz(context, 
                      HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      m, n, k,
                      descr, A->nnz,
                      A->d_rowIndices, A->d_colIndices, 
                      descr, B->nnz,
                      B->d_rowIndices, B->d_colIndices,
                      descr, C->d_rowIndices,
                      nnzTotalDevHostPtr
                      );

	if(status!=HIPSPARSE_STATUS_SUCCESS){
		std::cout<<"Error occured in finding NNZ";
		std::cout<<"\n Status "<<status;
		exit(0);
	}

	if(NULL != nnzTotalDevHostPtr){
		C->nnz = *nnzTotalDevHostPtr;
	}else{
		hipMemcpy(&C->nnz, C->d_rowIndices + m, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&base, C->d_rowIndices, sizeof(int), hipMemcpyDeviceToHost);
		C->nnz -= base;
	}

	allocate_csr_col_val(C, C->nnz);	

	hipsparseDcsrgemm(context, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m,n,k,
			descr, A->nnz,
			A->d_values, A->d_rowIndices, A->d_colIndices,
			descr, B->nnz,
			B->d_values, B->d_rowIndices, B->d_colIndices,
			descr, 
			C->d_values, C->d_rowIndices, C->d_colIndices);

	hipDeviceSynchronize();
}

int main (int argc, char *argv[] ){
	/**************
 	* NetMF small *
	**************/
	/* Argument order 
	1. Dataset name
	2. Window Size
	3. Dimension
	4. B
	5. Input
	6. Output
	7. Mapping file
	*/
	typedef std::chrono::high_resolution_clock Clock;
	typedef std::chrono::milliseconds milliseconds;
        Clock::time_point begin, end;
        Clock::time_point overall_begin, overall_end;
	info profile; 
	profile.dataset = argv[1];
	profile.algo = "small-dense";
	/* Section 0: Preliminaries */

	/* Settings */
	int window_size = std::atoi(argv[2]);
	int dimension = std::atoi(argv[3]);
	int b = std::atoi(argv[4]);

	profile.window_size = window_size;
	profile.dimension = dimension;

	/* Load graph */
        log("Reading data from file");
	
	//Graph g =  read_graph("../data/test/small_test.csv","edgelist");
	Graph g =  read_graph(argv[5],"edgelist", argv[7]);
	begin = Clock::now(); 
	//Graph g =  read_graph("../../nrl-data/wikipedia.edgelist","edgelist");
	end = Clock::now();

	profile.iptime = std::chrono::duration_cast<milliseconds>(end - begin);

	if(DEBUG){
		if(VERBOSE){
			log("Printing adj matrix");
			print_matrix(g.adj, g.size);
		}
	}	
	
	log("Printing degree matrix");
	if(DEBUG){
		if(VERBOSE){
			print_matrix(g.degree, g.size);
		}
	}


	/* CUDA housekeeping */
	begin = Clock::now();
	float num_threads = 128;
	dim3 threads(num_threads);
	dim3 grids((int)ceil((float)(g.size*g.size)/num_threads));

	/* CuSparse housekeeping */
	hipsparseHandle_t cusparse_handle;    
	hipsparseCreate(&cusparse_handle);	

	hipsparseMatDescr_t mat_descr;
	hipsparseCreateMatDescr(&mat_descr);
	hipsparseSetMatType(mat_descr, 
			HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(mat_descr, 
			HIPSPARSE_INDEX_BASE_ZERO);
	int LDA = g.size;

	/* CuBlas Housekeeping */
	log("Creating cuBlas variables");
	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	end = Clock::now();
	profile.init = std::chrono::duration_cast<milliseconds>(end - begin);


	/* Section 1. Move data to device */	

	/* Procedure 
	   1. Create dense adjacency and degree matrix on device
	   2. Allocate space for adjacency and degree matrix on device
	   3. Copy dense matrix from host to device
	   4. Preprocess degree and adjacency matrix for laplacian computation
	   5. Create CSR data structure for both matrices
	   6. Compute nnz/row of dense matrix
	   7. Apply Dense2CSR
	 */
	
	begin = Clock::now();
	/* Step 1: Create dense adjacency matrix and degree matrixx on device */
	log("Creating dense device array");
	double *adj_device_dense;	
	double *degree_device_dense; 
	//double *adj_host_dense;	
	//double *degree_host_dense; 

	/* Step 2a: Allocate space for adjacency and degree matrix on device*/
	log("Allocating space for degree and adjacency mat on device");
	hipMalloc(&adj_device_dense, 
			g.size * g.size * sizeof(double)); 	
	hipMalloc(&degree_device_dense, 
			g.size * sizeof(double)); 

	/* Step 2a: Allocate space for adjacency and degree matrix on host */
	log("Allocating space for degree and adjacency matrix on host");
	//adj_host_dense = (double *) malloc(g.size * g.size * sizeof(double));
	//degree_host_dense = (double *) malloc(g.size * sizeof(double));

	/* Step 3: Copy dense matrix from host to device */
	log("Copying dense matrix from host to device");	
	hipMemcpy(adj_device_dense, 
			g.adj, 
			g.size * g.size * sizeof(double), 
			hipMemcpyHostToDevice);	
	hipMemcpy(degree_device_dense, 
			g.degree1D, 
			g.size * sizeof(double), 
			hipMemcpyHostToDevice);

	/*REMOVE*/
	std::sort(g.degree1D,g.degree1D + g.size);

	/*Step 4: Compute volume and preprocess degree */
	preprocess_laplacian<<<grids,threads>>>(adj_device_dense, degree_device_dense, g.size);
	end = Clock::now();
	profile.gpuio = std::chrono::duration_cast<milliseconds>(end - begin);

	begin = Clock::now();
	log("Moved data from host to device");

	/* Section 2: Compute X = D^{-1/2} * A * D^{-1/2} */
	/* Procedure
	   1. Compute D' = D^{-1/2}
	   2. Compute X' = D' * A
	   3. Compute X = X' * D'
	*/
	
	/* Step 1: Compute D' = D^{-1/2} */
	log("Computing normalized D");
	compute_d<<<grids, threads>>>(degree_device_dense, g.size);
	hipDeviceSynchronize();
	end = Clock::now();
	profile.compute_d = std::chrono::duration_cast<milliseconds>(end - begin);

	log("Computed normalized D");
	overall_begin = Clock::now();
	begin = Clock::now();
	/* Step 2: Compute X' = D' * A */
	log("Computing X' = D' * A");
	double *X_temp_device;
	//double *X_temp_host;

	hipMalloc(&X_temp_device, g.size * g.size * sizeof(double));
	hipMemset(X_temp_device, 0, g.size * g.size);

	hipblasDdgmm(cublas_handle, HIPBLAS_SIDE_LEFT,
		g.size, g.size,
		adj_device_dense, g.size, 
		degree_device_dense, 1,
		X_temp_device, g.size);
	hipDeviceSynchronize();
	

//	/* Step 3: Compute X = X' * D */
	log("Computing X = X' * D");
	double *X_device;
	//double *X_host;
	hipMalloc(&X_device, g.size * g.size * sizeof(double));
	hipMemset(X_device, 0, g.size * g.size);

	hipblasDdgmm(cublas_handle, HIPBLAS_SIDE_RIGHT,
		g.size, g.size,
		X_temp_device, g.size, 
		degree_device_dense, 1,
		X_device, g.size);
	
	hipDeviceSynchronize();	
	hipFree(X_temp_device);
	hipFree(adj_device_dense);
	end = Clock::now();
	profile.compute_x = std::chrono::duration_cast<milliseconds>(end - begin);


	/* Section 3: Compute S = sum(X^{0}....X^{window_size}) */
	/* Procedure
	  1. Copy X to S
	  2. Copy X to W
	  3. W' = W * X
	  4. S' = S + W'  
	  5. W = W'
	  6. S = S'
	*/
	
	/* Step 0: Declare all variables */
	double *S_device;
        //double *S_host;
	double *W_device;
        //double *W_host;
	double *S_temp_device;
        //double *S_temp_host;
	double *W_temp_device;
        //double *W_temp_host;	

	const double alpha = 1.00;
	double beta = 1.00;

	begin = Clock::now();
	/* Step 1: Copy X to S */
	log("Copying X to S");

	hipMalloc(&S_temp_device, g.size * g.size * sizeof(double));
	hipMalloc(&S_device, g.size * g.size *sizeof(double));
	hipMemcpy(S_device, X_device, g.size * g.size * sizeof(double), hipMemcpyDeviceToDevice);

	/* Step 2: Copy X to temp */
	log("Copying X to W");
	
	hipMalloc(&W_temp_device, g.size * g.size * sizeof(double));
	hipMalloc(&W_device, g.size * g.size *sizeof(double));
	hipMemcpy(W_device, X_device, g.size * g.size * sizeof(double), hipMemcpyDeviceToDevice);

	for(int i=2;i<=window_size;i++){
		/* Step 3: temp' = temp * X */
		log("Computing W' = W * X");
		hipMemset(W_temp_device, 0, g.size * g.size);
		beta = 0;
		hipblasDgemm(cublas_handle, 
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				g.size, g.size, g.size,
				&alpha,
				W_device, LDA,
				X_device, LDA,
				&beta,
				W_temp_device, LDA);

		/* Step 4: S = S + temp */
		log("Computing S' = S + W'");
		hipMemset(S_temp_device, 0, g.size * g.size);
		beta = 1;
		hipblasDgeam(cublas_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				g.size, g.size,
				&alpha,
				S_device, LDA,
				&beta,
				W_temp_device, LDA,
				S_temp_device, LDA);

		/* Step 5: temp = temp' */
		log("Copying W' to W");
		hipMemset(W_device, 0, g.size * g.size);
		hipMemcpy(W_device, W_temp_device, g.size * g.size * sizeof(double), hipMemcpyDeviceToDevice);

		/* Step 6: S = S' */
		log("Copying S' to S");
		hipMemset(S_device, 0, g.size * g.size);
		hipMemcpy(S_device, S_temp_device, g.size * g.size * sizeof(double), hipMemcpyDeviceToDevice);
	}

	hipFree(S_temp_device);
	hipFree(W_temp_device);
	hipFree(W_device);

        /* Section 4: Compute S = S * (vol / (window_size * b)) */

	/* Procedure 
	   1. Compute val = vol / (window_size * b)
	   2. Compute S[i] = S[i] / val
	*/

	log("Applying Transformation on S");
	/* Step 1: Compute val = vol / (window_size * b) */
	const double val = ((double) g.volume)/(((double) window_size) * ((double) b));

	if(DEBUG){
		std::cout<<"Mult value"<<val<<std::endl;
	}

	/* Step 2: Compute S[i] = S[i] * val */

	hipblasDscal(cublas_handle, g.size * g.size,
                    &val,
                    S_device, 1);

	//S_host = (double *) malloc(g.size * g.size * sizeof(double));
	end = Clock::now();
	profile.compute_s = std::chrono::duration_cast<milliseconds>(end - begin);

	begin = Clock::now();
	log("Computing M");

        /* Section 5: Compute M = D^{-1/2} * S * D^{-1/2} */
	/* Procedure
	   1. Compute M' = D' * S
	   2. Compute M = M' * D'
	*/

	/* Step 1: Compute M' = D' * S */
	log("Computing M' = D' * S");

	double *M_temp_device;
	//double *M_host;

	hipMalloc(&M_temp_device, g.size * g.size * sizeof(double));

	hipblasDdgmm(cublas_handle, HIPBLAS_SIDE_LEFT,
		g.size, g.size,
		S_device, g.size, 
		degree_device_dense, 1,
		M_temp_device, g.size);

	hipFree(S_device);

	log("Computing M = M' * S'");
	double *M_device;
	hipMalloc(&M_device, g.size * g.size * sizeof(double));

	hipblasDdgmm(cublas_handle, HIPBLAS_SIDE_RIGHT,
		g.size, g.size,
		M_temp_device, g.size, 
		degree_device_dense, 1,
		M_device, g.size);

	hipFree(M_temp_device);

	/* Section 6: Compute M'' = log(max(M,1)) */
	
	/* Procedure 
	   1. Prune M and take log
	   2. Create CSR struct for M''
	   3. Compute nnzPerVector for M''
	*/

	/* Step 1: Prune M and take log */
	log("Pruning M");

	prune_m<<<grids,threads>>>(M_device, g.size);
       	hipDeviceSynchronize(); 

	log("Pruned M");

	/* Step 2: Create CSR struct for both matrices */
	log("Converting dense matrix to CSR format");	
	csr M_cap;    /* Variable to hold adjacency matrix in CSR format */

	M_cap.nnz = 0; /* Initialize number of non zeros in adjacency matrix */

	/* Step 6: Compute nnz/row of dense matrix */	
	log("Computing nnzPerVector for M''");

	hipMalloc(&M_cap.d_nnzPerVector, 
			g.size * sizeof(int));
	hipsparseDnnz(cusparse_handle, 
			HIPSPARSE_DIRECTION_ROW, 
			g.size, g.size, 
			mat_descr, 
			M_device, LDA, 
			M_cap.d_nnzPerVector, &M_cap.nnz);
	M_cap.h_nnzPerVector = (int *)malloc(g.size * sizeof(int));
	hipMemcpy(M_cap.h_nnzPerVector, 
			M_cap.d_nnzPerVector, 
			g.size * sizeof(int), 
			hipMemcpyDeviceToHost); 
	if(DEBUG){
    		printf("Number of nonzero elements in dense adjacency matrix = %i\n", M_cap.nnz);
    		
		if(VERBOSE)
		for (int i = 0; i < g.size; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, M_cap.h_nnzPerVector[i]);
	}


	/* Step 6: Convert dense matrix to sparse matrices */
	allocate_csr(&M_cap, M_cap.nnz, g.size);
	hipsparseDdense2csr(cusparse_handle, 
			g.size, g.size, 
			mat_descr,
		       	M_device,	
			LDA, 
			M_cap.d_nnzPerVector, 
			M_cap.d_values, M_cap.d_rowIndices, M_cap.d_colIndices); 
	if(VERBOSE){
		device2host(&M_cap, M_cap.nnz, g.size);	
		print_csr(
    			g.size,
    			M_cap.nnz,
    			M_cap,
    			"Adjacency matrix");
	}

	hipFree(M_device);

	device2host(&M_cap, M_cap.nnz, g.size);
	log("Completed conversion of data from dense to sparse");
	end = Clock::now();
	profile.compute_m = std::chrono::duration_cast<milliseconds>(end - begin);
		
	/* Section 7: Compute SVD of objective matrix */	

	begin = Clock::now();
	char whichS = 'L';
	char whichV = 'L';

	MKL_INT pm[128];
	mkl_sparse_ee_init(pm);
	//pm[1] = 100;
	//pm[2] = 2;
	//pm[4] = 60;

	MKL_INT mkl_rows = g.size;
	MKL_INT mkl_cols = g.size;


	//MKL_INT rows_start[mkl_rows];
	//MKL_INT rows_end[mkl_rows];

	MKL_INT *rows_start;
	MKL_INT *rows_end;

	rows_start = (MKL_INT *)mkl_malloc(mkl_rows * sizeof(MKL_INT),64);
	rows_end = (MKL_INT *)mkl_malloc(mkl_rows * sizeof(MKL_INT),64);

	for(int i=0;i<mkl_rows;i++){
		rows_start[i] = M_cap.h_rowIndices[i];
		rows_end[i] = M_cap.h_rowIndices[i+1];
	}

	
	//MKL_INT mkl_col_idx[M_cap.nnz];

	MKL_INT *mkl_col_idx;
	mkl_col_idx = (MKL_INT*)mkl_malloc(M_cap.nnz * sizeof(MKL_INT), 64);

	int mkl_temp;
	for(int i=0;i<M_cap.nnz;i++){
		mkl_temp = M_cap.h_colIndices[i];
		mkl_col_idx[i] = mkl_temp;
	}


	sparse_matrix_t M_mkl;
	sparse_index_base_t indexing = SPARSE_INDEX_BASE_ZERO;

	mkl_sparse_d_create_csr(&M_mkl, indexing,
					mkl_rows, mkl_cols,
					rows_start, rows_end,
					mkl_col_idx, M_cap.h_values);

	log("Created MKL sparse");

	matrix_descr mkl_descrM;
	mkl_descrM.type = SPARSE_MATRIX_TYPE_GENERAL;	
	mkl_descrM.mode = SPARSE_FILL_MODE_UPPER;
	mkl_descrM.diag = SPARSE_DIAG_NON_UNIT;

	MKL_INT k0 = dimension;
	MKL_INT k;

	double *E_mkl, *K_L_mkl, *K_R_mkl, *res_mkl;

	E_mkl = (double *)mkl_malloc(k0 * sizeof(double), 128);
	K_L_mkl = (double *)mkl_malloc( k0*mkl_rows*sizeof( double), 128 );
        K_R_mkl = (double *)mkl_malloc( k0*mkl_cols*sizeof( double), 128 );
        res_mkl = (double *)mkl_malloc( k0*sizeof( double), 128 );

	memset(E_mkl, 0 , k0);
	memset(K_L_mkl, 0 , k0);
	memset(K_R_mkl, 0 , k0);
	memset(res_mkl, 0 , k0);

	int mkl_status = 0;

	log("Computing SVD via MKL");
	mkl_status = mkl_sparse_d_svd(&whichS, &whichV, pm,
			M_mkl, mkl_descrM,
			k0, &k,
			E_mkl,
			K_L_mkl,
			K_R_mkl,
			res_mkl);
	
	if(mkl_status){
		std::cout<<"SVD failed"<<std::endl;
		exit(0);	
	}


	log("Computed SVD via MKL");

	if(DEBUG){
	std::cout<<"Number of singular found: "<<k<<std::endl;
	for(int i=0;i<k0;i++){ std::cout<<E_mkl[i]<<" ";} std::cout<<"\n";
	}

	double *U_device, *Si_device;
	//double *U_host;
	double *Si_host;
	double *E_device, *E_host;

	hipMalloc(&U_device, g.size * dimension * sizeof(double));
	hipMalloc(&E_device, g.size * dimension * sizeof(double));
	hipMalloc(&Si_device, dimension * sizeof(double));

	//U_host = (double *) malloc(g.size * dimension * sizeof(double));
	E_host = (double *) malloc(g.size * dimension * sizeof(double));
	Si_host = (double *) malloc(dimension * sizeof(double));

	hipMemcpy(U_device, K_L_mkl, g.size * dimension * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Si_device, E_mkl, dimension * sizeof(double), hipMemcpyHostToDevice);

	transform_si<<<grids,threads>>>(Si_device, dimension);

	hipMemcpy(Si_host, Si_device, dimension * sizeof(double), hipMemcpyDeviceToHost);

	std::cout<<"\n";
	hipblasDdgmm(cublas_handle, HIPBLAS_SIDE_RIGHT,
		g.size, dimension,
		U_device, g.size, 
		Si_device, 1.0,
		E_device, g.size);

	hipMemcpy(E_host, E_device, g.size * dimension * sizeof(double), hipMemcpyDeviceToHost);

	end = Clock::now();
	profile.svd = std::chrono::duration_cast<milliseconds>(end - begin);


	overall_end = Clock::now();
	profile.emb = std::chrono::duration_cast<milliseconds>(overall_end - overall_begin);


	write_profile("profile.txt", profile);
	write_embeddings(argv[6],E_host, g.size, dimension);

	mkl_free(rows_start);	
	mkl_free(rows_end);	
	mkl_free(mkl_col_idx);	

}
