#include "hip/hip_runtime.h"
/* 
TODO: 
1. Change thread architecture
2. Use cuBlas for addition
3. Async copy to device
*/
/*
1. Questions for prof.
In kernel filter_E, is it possible to copy without a kernel?
*/
#include<stdlib.h>
#include<iostream>
#include<time.h>
#include<chrono>

#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>

#include "../utils/graph.h"
#include "../utils/io.h"

__global__ void compute_d(float* deg, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= size) return;

	id = id * size + id;
	
	/* Make assumption here that graph is  	*/
	/* connected and every node has degree 	*/
        /* atleast 1. 		       		*/

	deg[id] = sqrt(1/deg[id]); 
}

// Make this kernel 2D
__global__ void compute_s(float* S, float* X, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] += X[id]; 
}

// Make this kernel 2D
__global__ void transform_s(float* S, int volume, int window_size, int b, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] = (S[id] * float(volume))/ ((float) window_size * (float) b); 
}

//Make this kernel 2D
__global__ void transform_m(float* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	M[id] =logf(M[id] > 1?M[id]:1);
}

__global__ void sqrt_si(float* S, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = sqrt((float) S[id]);
}

__global__ void filter_e(float *W, float *e, int size, int window_size, int rank){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= rank) return;

	float el = W[size - rank + id];
	float val = 0;

	if(el >= 1)
		e[id] = 1;
	else{
	      val = (el * (1 - powf(el, window_size))) / ((1-el) * window_size);
	      e[id] = 0 > val ? 0: val;
	}
	e[id] = sqrt(e[id]);

}

// Make this kernel 2D
__global__ void filter_E(float *X, float *E, int size, int rank){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id>=rank) return;
	
	for(int i=0;i<size;i++){
		E[id * size + i] = X[(size - rank + id) * size + i];
	}
		
	
}

void print_matrix(float* S, int size){
	std::cout<<std::endl<<std::endl;
	for(int i=0;i<size;i++){
		for(int j=0;j<size;j++){
			std::cout<<S[i*size + j]<<" ";
		}
		std::cout<<std::endl;
	}

}


int main ( void ){
	/**************
 	* NetMF large *
	**************/

	/* General housekeeping */
	typedef std::chrono::high_resolution_clock Clock;
	typedef std::chrono::milliseconds milliseconds;
        Clock::time_point begin, end;
	info profile; 
	profile.dataset = "blogcatalog";
	profile.algo = "large";

	/* Load graph */
        log("Reading data from file");

        begin = Clock::now(); 
	Graph g =  read_graph("../data/blogcatalog/edges.csv","edgelist");
        end = Clock::now(); 

	profile.iptime = std::chrono::duration_cast<milliseconds>(end - begin);	

	/* CUDA housekeeping */
	log("Running Initialization routine");
	log("Defining Threads");
	begin = Clock::now();
	float num_threads = 128;	
	dim3 threads(num_threads);
	dim3 grid((int)ceil((float)g.size/num_threads));

	/* cuBlas housekeeping */
	log("Creating cuBlas variables");
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float al=1.0f;
	float bet=1.0f;

	/* cuSolver housekeeping */
	log("Setting up cuSolver");	
	int lwork = 0;
	float *d_work, *d_rwork;
	int *devInfo;
	signed char jobu = 'A';
	signed char jobvt = 'N';
	
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipMalloc(&devInfo, sizeof(int));
	hipsolverHandle_t cusolverH;
	hipsolverDnCreate(&cusolverH);

	/* Initialize and allocate variables */
	log("Setting up host variables");
	int window_size = 10;
	profile.window_size = window_size;
	int b = 1;
	int rank = 2;
	int dimension = 2;
	const float scale = float(g.volume)/float(b);
	int size = g.size * g.size * sizeof(float);
	profile.dimension = dimension;
	
	float *X;
	float *MMT;
	float *Embedding;

	X = (float *)malloc(size);
	MMT = (float *) malloc(size);
	Embedding = (float *)malloc(g.size * dimension * sizeof(float));
	
	memset(MMT, 0, size);
	memset(X, 0, size);
	memset(Embedding, 0, g.size * dimension *sizeof(float));
	
	log("Setting up device variables");
	float *D_device;
	float *A_device;
	float *temp_device;
	float *X_device;
	float *M_device;
	float *U_device, *VT_device, *Si_device;
	float *W_device;
	float *e_device;
	float *E_device;
	float *MMT_device;
	float *Embedding_device;
	
	hipMalloc((void **)&Embedding_device, g.size * dimension *sizeof(float));
	hipMalloc((void **)&MMT_device, size);
	hipMalloc((void **)&W_device, sizeof(float) * g.size);
	hipMalloc((void **)&e_device, sizeof(float) * rank);
	hipMalloc((void **)&E_device, sizeof(float) * g.size * rank);
	hipMalloc((void**)&U_device, size);
	hipMalloc((void**)&Si_device, g.size * sizeof(float));
	hipMalloc((void**)&VT_device, size);
	hipMalloc((void**)&D_device, size);
	hipMalloc((void**)&A_device, size);
	hipMalloc((void**)&X_device, size);
	hipMalloc((void**)&temp_device, size);
	hipMalloc((void**)&M_device, size);

	end = Clock::now();
	profile.init = std::chrono::duration_cast<milliseconds>(end - begin);
	
	/* Copy necessary variables to device */
	/* 
	   Note: Make this a non-blocking operation using
	   using Async since g.degree, g.adj and g.size 
	   are available at the very beginning
	*/
	log("Moving data to device");
	begin = Clock::now();
	hipMemcpy(D_device, g.degree, size, hipMemcpyHostToDevice);	
	hipMemcpy(A_device, g.adj, size , hipMemcpyHostToDevice);	
	end = Clock::now();
	profile.gpuio = std::chrono::duration_cast<milliseconds>(end - begin);;

	/* Compute D = D^{-1/2} */
	begin = Clock::now();
	log("Computing normalized D");
	compute_d<<<grid, threads>>>(D_device, g.size);
	hipDeviceSynchronize();
	end = Clock::now();
	profile.compute_d = std::chrono::duration_cast<milliseconds>(end - begin);;

	/* Compute X = D^{-1/2}AD^{-1/2} */
	log("Computing X");
	begin = Clock::now();
	hipblasSgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
	            A_device,g.size, 
		    D_device, g.size,
		    &bet, 
		    temp_device, g.size);
	hipDeviceSynchronize();	
	hipblasSgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
		    D_device, g.size,
	            temp_device,g.size, 
		    &bet, 
		    X_device, g.size);
	hipDeviceSynchronize();	
	end = Clock::now();
	profile.compute_x = std::chrono::duration_cast<milliseconds>(end - begin);;	

	/* Eigen decomposition of X */
	log("Eigen Decomposition of X");

	hipsolverDnSsyevd_bufferSize(cusolverH,jobz, uplo, g.size, X_device, g.size, W_device, &lwork);
	hipMalloc(&d_work, sizeof(float) * lwork);

	begin = Clock::now();
	hipsolverDnSsyevd(cusolverH, 
			jobz, uplo, g.size, 
			X_device, g.size, 
			W_device, d_work, 
			lwork, devInfo);
	
	hipDeviceSynchronize();
	end = Clock::now();
	profile.compute_s = std::chrono::duration_cast<milliseconds>(end - begin);;	

	begin = Clock::now();
	log("Filtering eigenvalues and eigen vectors");
	filter_e<<<grid, threads>>>(W_device, e_device, g.size, window_size, rank);
	hipDeviceSynchronize();	
	
	filter_E<<<grid, threads>>>(X_device, E_device, g.size, rank);	
	hipDeviceSynchronize();	

	hipMemset(temp_device, 0, g.size * g.size * sizeof(float));
	
	hipblasSdgmm(handle,
		    HIPBLAS_SIDE_LEFT,
		    g.size, rank,
		    E_device, g.size,
		    D_device, g.size + 1,
		    temp_device, g.size);
	hipDeviceSynchronize();	

	hipblasSdgmm(handle,
		    HIPBLAS_SIDE_RIGHT,
		    g.size, rank,
		    temp_device, g.size,
		    e_device, 1,
		    M_device, g.size);
	hipDeviceSynchronize();	

	hipblasSgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_T, 
		    g.size, g.size, g.size,
		    &al,
	            M_device,g.size, 
		    M_device, g.size,
		    &bet, 
		    MMT_device, g.size);

	hipDeviceSynchronize();	
	hipblasSscal(handle, g.size * g.size,
			&scale,
			MMT_device, 
			1);
	hipDeviceSynchronize();	
	
	transform_m<<<grid,threads>>>(MMT_device, g.size);
	hipDeviceSynchronize();	
	end = Clock::now();
	profile.compute_m = std::chrono::duration_cast<milliseconds>(end - begin);	

	begin = Clock::now();	
	hipsolverDnSgesvd(cusolverH, jobu, jobvt, 
			g.size, g.size, MMT_device, g.size, 
			Si_device, 
			U_device, g.size, 
			VT_device, g.size, 
			d_work, 
			lwork, 
			d_rwork, 
			devInfo); 
	hipDeviceSynchronize();	
	end = Clock::now();
	profile.svd = std::chrono::duration_cast<milliseconds>(end - begin);	

	begin = Clock::now();
	sqrt_si<<<grid, threads>>>(Si_device, dimension);
	hipDeviceSynchronize();	
	hipblasSdgmm(handle, 
	    HIPBLAS_SIDE_RIGHT, 
	    g.size, dimension,
	    U_device, g.size,
	    Si_device,-1, 
	    Embedding_device, g.size);
		
        hipDeviceSynchronize();
	

	hipMemcpy(Embedding, Embedding_device,sizeof(float)* g.size * dimension, hipMemcpyDeviceToHost);
	end = Clock::now();
	profile.emb = std::chrono::duration_cast<milliseconds>(end - begin);	

	write_embeddings("blogcatalog.emb",Embedding, g.size, dimension);	
	write_profile("profile.txt", profile);		
	log("Done");
	/***********
	* Clean up *
	***********/

	free(X);

	// DEVICE
	hipFree(D_device);
	hipFree(A_device);
	hipFree(X_device);
	hipFree(temp_device); 

	hipFree(d_work);
	hipFree(d_rwork);
	hipFree(devInfo);

}
