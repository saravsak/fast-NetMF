#include "hip/hip_runtime.h"
/* 
TODO: 
1. Change all doubles to float 
2. USe cuBlas for addition
*/

/* 
Question for prof
1. Copy stuff within GPU
2. Launch kernel without copy
3. Is it better to do more redundant work in one thread or one more kernle to do it once?
4. Results are wrong if I use same variable as result. Why?
*/
#include<stdlib.h>
#include<iostream>

#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>

#include "../utils/graph.h"
#include "../utils/graphio.h"

__global__ void compute_d(double* deg, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= size) return;

	id = id * size + id;
	
	// Make assumption here that graph is connected and every node has degree atleast 1.
	deg[id] = sqrt(1/deg[id]); 
}

__global__ void compute_s(double* S, double* X, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] += X[id]; 
}

__global__ void transform_s(double* S, int volume, int window_size, int b, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] = (S[id] * float(volume))/ ((float) window_size * (float) b); 
}

__global__ void transform_m(double* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	M[id] =logf(M[id] > 1?M[id]:1);
}

__global__ void sqrt_si(double* S, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = sqrt((float) S[id]);
}


int main ( void ){

	/**************
 	* NetMF small *
	**************/
	
	/* Load graph */
	Graph g =  read_graph("../data/test/small_test.csv","edgelist");

	/* CUDA housekeeping */
	dim3 threads(128);
	dim3 grid((int)ceil((float)g.size/128));

	/* cuBlas housekeeping */	
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	double al=1.0f;
	double bet=1.0f;

	/* Initialize and allocate variables */
	// HOST
	double *X;
	double *S;
	double *M;
	
	int window_size = 10;
	int size = g.size * g.size * sizeof(double);
	int b = 1;
	int dimension = 2;
	
	X = (double *)malloc(size);
	S = (double *)malloc(size);
	M = (double *)malloc(size);	

	// DEVICE
	double *D_device;
	double *temp_device, *temp1_device, *X_device;
	double *A_device;
	double *S_device;
	double *M_device;

	hipMalloc(&D_device, size);
	hipMalloc(&A_device, size);
	hipMalloc(&X_device, size);
	hipMalloc(&temp_device, size);
	hipMalloc(&temp1_device, size);
	hipMalloc(&S_device, size);
	hipMalloc(&M_device, size);

	hipMemset(A_device, 0, size);
	hipMemset(D_device, 0, size);
	hipMemset(X_device, 0, size);
	hipMemset(S_device, 0, size);
	hipMemset(M_device, 0, size);
	hipMemset(temp_device, 0, size);
	hipMemset(temp1_device, 0, size);

	/* Copy necessary variables to device */
	hipMemcpy(D_device, g.degree, size, hipMemcpyHostToDevice);	
	hipMemcpy(A_device, g.adj, size , hipMemcpyHostToDevice);	

	/* Compute D = D^{-1/2} */
	compute_d<<<grid, threads>>>(D_device, g.size);
	hipDeviceSynchronize();

	/* Compute X = D^{-1/2}AD^{-1/2} */

	hipblasDgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
	            A_device,g.size, 
		    D_device, g.size,
		    &bet, 
		    temp_device, g.size);
	
	hipblasDgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
		    D_device, g.size,
	            temp_device,g.size, 
		    &bet, 
		    X_device, g.size);
	
	/* Compute S = sum(X^{0}....X^{window_size}) */
	
	// This might be too slow. Experiment to see if you can use a custom kernel 
	hipMemcpy(S_device, X_device, size, hipMemcpyDeviceToDevice);
	hipMemcpy(temp_device, X_device, size, hipMemcpyDeviceToDevice);

	for(int i=2;i<=window_size;i++){
		hipblasDgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
		    X_device, g.size,
	            temp_device,g.size, 
		    &bet, 
		    temp1_device, g.size);
		
		// Use cublas addition functions
		compute_s<<<grid, threads>>>(S_device, temp1_device, g.size);
		hipMemcpy(temp_device, temp1_device, size, hipMemcpyDeviceToDevice);
		hipMemset(temp1_device,0,size);
	}

	// Compute S = S * (vol / (window_size * b))
	transform_s<<<grid,threads>>>(S_device,g.volume, window_size, b, g.size);
	hipMemcpy(S, S_device, size, hipMemcpyDeviceToHost);
	
	std::cout<<std::endl<<std::endl;
	for(int i=0;i<g.size;i++){
		for(int j=0;j<g.size;j++){
			std::cout<<S[i*g.size + j]<<" ";
		}
		std::cout<<std::endl;
	}
	
	// Compute M = D^{-1/2} * S * D^{-1/2}
	hipMemset(temp_device, 0, size); 

	hipblasDgemm(handle, 
	    HIPBLAS_OP_N, HIPBLAS_OP_N, 
	    g.size, g.size, g.size,
	    &al,
	    S_device, g.size,
	    D_device,g.size, 
	    &bet, 
	    temp_device, g.size);

	hipblasDgemm(handle, 
	    HIPBLAS_OP_N, HIPBLAS_OP_N, 
	    g.size, g.size, g.size,
	    &al,
	    D_device, g.size,
	    temp_device,g.size, 
	    &bet, 
	    M_device, g.size);
		
	// Compute M = log(max(Mi,1))
	transform_m<<<grid,threads>>>(M_device, g.size);
	hipMemcpy(M, M_device, size, hipMemcpyDeviceToHost);

	// Perform SVD on M
	double *U, *VT, *Si;
	U = (double*)malloc(size);
	VT = (double*)malloc(size);
	Si = (double*)malloc(g.size * sizeof(double));

	double *U_device, *VT_device, *Si_device;
	double *W_device; //auxillary device array

	hipsolverHandle_t cusolverH;


	hipMalloc(&U_device, size);
	hipMalloc(&Si_device, g.size * sizeof(double));
	hipMalloc(&VT_device, size);
	hipMalloc(&W_device, size);

	int lwork = 0;
	double *d_work, *d_rwork;

	hipsolverDnDgesvd_bufferSize(cusolverH,g.size, g.size,&lwork);
	hipMalloc(&d_work, sizeof(double) * lwork);

	signed char jobu = 'A';
	signed char jobvt = 'A';
	int *devInfo;

	hipsolverDnDgesvd(cusolverH, jobu, jobvt, 
			g.size, g.size, A_device, 
			g.size, Si_device, 
			U_device, g.size, 
			VT_device, g.size, 
			d_work, lwork, d_rwork, devInfo); 
	
	hipDeviceSynchronize();
	
	// TODO: Clip vector to be of dimension D.

	sqrt_si<<<grid, threads>>>(Si_device, g.size);	
	hipblasDdgmm(handle, 
	    HIPBLAS_SIDE_LEFT, 
	    g.size, g.size,
	    U_device, g.size,
	    Si_device,1, 
	    W_device,  g.size);
	
	
	/***********
	* Clean up *
	***********/

	//free(g);

	//hipFree(D_device);

	// Function to print matrix
/*
	std::cout<<std::endl<<std::endl;
	for(int i=0;i<g.size;i++){
		for(int j=0;j<g.size;j++){
			std::cout<<S[i*g.size + j]<<" ";
		}
		std::cout<<std::endl;
	}
*/
}
