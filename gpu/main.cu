#include "hip/hip_runtime.h"
// Questionf for prof
// 1. Do we need to store in column major order for cuBlas
// 2. Do we need to copy everytime or once is enough
// 3. Ask if DAD calculation is correct

# include<stdlib.h>
# include<hip/hip_runtime.h>
# include<hipblas.h>

#include "../utils/graph.h"
#include "../utils/graphio.h"

#include<iostream>

__global__ void compute_d(double* deg, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;


	if(id >= size) return;

	id = id * size + id;
	
	// Make assumption here that graph is connected and every node has degree atleast 1.
	deg[id] = sqrt(1/deg[id]); 
}

int main ( void ){

	/**************
 	* NetMF small *
	**************/

	/* Load graph */
	Graph g =  read_graph("../data/test/small_test.csv","edgelist");
	std::cout<<"Printing i/p graph"<<std::endl;
	g.print_graph();
	std::cout<<std::endl;
	std::cout<<std::endl;
	std::cout<<"Printing degree matrix"<<std::endl;
	g.print_degree();


	// Initialize all host and device variables
	double *D_device;
	double *temp_device, *X, *X_device;
	double *A_device;
	
	X = (double *)malloc(g.size * g.size * sizeof(double));
	
	hipMalloc(&D_device, g.size * g.size * sizeof(double));
	hipMalloc(&X_device, g.size * g.size * sizeof(double));
	hipMalloc(&A_device, g.size * g.size * sizeof(double));
	hipMalloc(&temp_device, g.size * g.size * sizeof(double));

	hipMemset(X_device, 0, g.size * g.size * sizeof(double));
	hipMemset(temp_device, 0, g.size * g.size * sizeof(double));

	/* Compute D = D^{-1/2} */
	// Create and allocate device variable to hold degree matrix

	// Copy degree matrix to device
	hipMemcpy(D_device, g.degree, g.size * g.size * sizeof(double), hipMemcpyHostToDevice);	
	
	// Call GPU Kernel

	std::cout<<"number of grids: "<<(int)ceil((float)g.size/128)<<std::endl;	

	dim3 threads(128);
	dim3 grid((int)ceil((float)g.size/128));
	compute_d<<<grid, threads>>>(D_device, g.size);

	// Copy from device to host
	hipMemcpy(g.degree, D_device, g.size * g.size * sizeof(double), hipMemcpyDeviceToHost);

	// Print results
	std::cout<<std::endl;
	std::cout<<std::endl;
	std::cout<<"Printing D^{-1/2}"<<std::endl;
	g.print_degree();

	/* Compute X = D^{-1/2}AD^{-1/2} */
	// Declare and allocate X matrix


	// Copy A and D to device 
	hipMemcpy(D_device, g.degree, g.size * g.size * sizeof(double), hipMemcpyHostToDevice);	
	hipMemcpy(A_device, g.adj, g.size * g.size * sizeof(double), hipMemcpyHostToDevice);	
	
	// Compute product on GPU
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	double al=1.0f;
	double bet=1.0f;
	int size = g.size;

	/* NOTE: cuBlas takes matrices in col major order */
	/* So instead of X = D^{-1/2}AD^{-1/2} we do      */
	/* X = AD^{-1/2} -> X = X             */

	hipblasDgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    size, size, size,
		    &al,
	            A_device,size, 
		    D_device, size,
		    &bet, 
		    temp_device, size);
	
	hipblasDgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    size, size, size,
		    &al,
		    D_device, size,
	            temp_device,size, 
		    &bet, 
		    X_device, size);

	// Copy X to host
	hipMemcpy(X, X_device, g.size * g.size * sizeof(double), hipMemcpyDeviceToHost);
	
	// Print X
	std::cout<<std::endl;
	std::cout<<std::endl;
	std::cout<<"Printing X"<<std::endl;
	for(int i=0;i<g.size;i++){
		for(int j=0;j<g.size;j++){
			std::cout<<X[i*g.size + j]<<" ";
		}
		std::cout<<std::endl;
	}
	
	/* Compute S = sum(X^{0}....X^{window_size}) */
		
	// Compute S = S * (vol / (window_size * b))
	
	// Compute M = D^{-1/2} * S * D^{-1/2}
	
	// Compute M = log(max(Mi,1))

	/*****************
 	*  CUBLAS SAMPLE *
	*****************/

//	hipError_t cudaStat;
//	hipblasStatus_t stat;
//	hipblasHandle_t handle;
//	int j;
//	float *x;
//	x = (float*) malloc(n*sizeof(*x));
//	for(j=0;j<n;j++)
//		x[j] = (float) j;
//
//	float *d_x;
//
//	cudaStat = hipMalloc((void**)&d_x, n*sizeof(*x));
//
//	stat = hipblasCreate(&handle);
//
//	stat = hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1);
//	int result;
//
//	stat = hipblasIsamax(handle, n, d_x, 1, &result);
//
//	std::cout<<x[result-1]<<std::endl;
//
//	hipFree(d_x);
//	hipblasDestroy(handle);
//	free(x);
//	return EXIT_SUCCESS;
//

	/***********
	* Clean up *
	***********/

	//free(g);

	//hipFree(D_device);
}
