#include "hip/hip_runtime.h"
/* 
TODO: 
1. Change thread architecture
2. Use cuBlas for addition
3. Async copy to device
*/

#include<stdlib.h>
#include<iostream>
#include<time.h>
#include<chrono>
#include<algorithm>

#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>

#include "../utils/graph.h"
#include "../utils/io.h"

#include "../lib/RedSVD-h"
#include<Eigen/Core>
#include<Eigen/Dense>

__global__ void compute_d(float* deg, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= size) return;

	id = id * size + id;
	
	/* Make assumption here that graph is  	*/
	/* connected and every node has degree 	*/
        /* atleast 1. 		       		*/

	deg[id] = sqrt(1/deg[id]); 
}

__global__ void compute_s(float* S, float* X, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] += X[id]; 
}

__global__ void transform_s(float* S, int volume, int window_size, int b, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	S[id] = (S[id] * float(volume))/ ((float) window_size * (float) b); 
}

__global__ void transform_m(float* M, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size * size) return;
	
	M[id] =logf(M[id] > 1?M[id]:1);
}

__global__ void sqrt_si(float* S, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id >= size) return;
	
	S[id] = sqrt((float) S[id]);
}

void print_matrix(float* S, int size){
	std::cout<<std::endl<<std::endl;
	for(int i=0;i<size;i++){
		for(int j=0;j<size;j++){
			std::cout<<S[i*size + j]<<" ";
		}
		std::cout<<std::endl;
	}

}


int main ( void ){
	/**************
 	* NetMF small *
	**************/

	/* General housekeeping */
	typedef std::chrono::high_resolution_clock Clock;
	typedef std::chrono::milliseconds milliseconds;
        Clock::time_point begin, end;
	info profile; 
	profile.dataset = "blogcatalog";
	profile.algo = "small";

	/* Load graph */
        log("Reading data from file");

        begin = Clock::now(); 
	Graph g =  read_graph("../data/test/small_test.csv","edgelist");
        end = Clock::now(); 

	profile.iptime = std::chrono::duration_cast<milliseconds>(end - begin);	

	/* CUDA housekeeping */
	log("Running Initialization routine");
	log("Defining Threads");

	begin = Clock::now();
	float num_threads = 128;	
	dim3 threads(num_threads);
	dim3 grid((int)ceil((float)g.size/num_threads));

	/* cuBlas housekeeping */
	log("Creating cuBlas variables");
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float al=1.0f;
	float bet=1.0f;

	/* cuSolver housekeeping */
	log("Setting up cuSolver");	
	int lwork = 0;
	signed char jobu = 'A';
	signed char jobvt = 'N';
	float *d_work, *d_rwork;
	int *devInfo;
	
	hipMalloc(&devInfo, sizeof(int));
	hipsolverHandle_t cusolverH;
	hipsolverDnCreate(&cusolverH);
	hipsolverDnSgesvd_bufferSize(cusolverH,g.size, g.size,&lwork);
	hipMalloc(&d_work, sizeof(float) * lwork);
	hipMalloc(&d_rwork, sizeof(float) *( g.size - 1));

	/* Initialize and allocate variables */
	log("Setting up host variables");
	float *U, *VT, *Si, *W;
	float *X;
	float *S;
	float *M;
	
	int window_size = 3;
	profile.window_size = window_size;
	int size = g.size * g.size * sizeof(float);
	int b = 1;
	int dimension = 3;
	profile.dimension = dimension;
	//int *devInfoH;
	
	X = (float *)malloc(size);
	S = (float *)malloc(size);
	M = (float *)malloc(size);	
	U = (float*)malloc(g.size * dimension *sizeof(float));
	VT = (float*)malloc(size);
	W = (float*)malloc(size);
	Si = (float*)malloc(g.size * sizeof(float));
	//devInfoH = (int *)malloc(sizeof(int));

	log("Setting up device variables");
	float *D_device;
	float *temp_device;
	float *temp1_device;
	float *X_device;
	float *A_device;
	float *S_device;
	float *M_device;
	float *M_temp_device;
	float *U_device, *VT_device, *Si_device;
	float *W_device; //auxillary device array

	hipMalloc(&U_device, g.size * dimension *sizeof(float));
	hipMalloc(&Si_device, g.size * sizeof(float));
	hipMalloc(&VT_device, size);
	hipMalloc(&W_device, size);

	hipMalloc(&D_device, size);
	hipMalloc(&A_device, size);
	hipMalloc(&X_device, size);
	hipMalloc(&temp_device, size);
	hipMalloc(&temp1_device, size);
	hipMalloc(&S_device, size);
	hipMalloc(&M_device, size);
	hipMalloc(&M_temp_device, size);

	hipMemset(A_device, 0, size);
	hipMemset(D_device, 0, size);
	hipMemset(X_device, 0, size);
	hipMemset(S_device, 0, size);
	hipMemset(M_device, 0, size);
	hipMemset(M_temp_device, 0, size);
	hipMemset(temp_device, 0, size);
	hipMemset(temp1_device, 0, size);
	end = Clock::now();
	profile.init = std::chrono::duration_cast<milliseconds>(end - begin);
	
	/* Copy necessary variables to device */
	/* 
	   Note: Make this a non-blocking operation using
	   using Async since g.degree, g.adj and g.size 
	   are available at the very beginning
	*/
	log("Moving data to device");
	begin = Clock::now();
	hipMemcpy(D_device, g.degree, size, hipMemcpyHostToDevice);	
	hipMemcpy(A_device, g.adj, size , hipMemcpyHostToDevice);	
	end = Clock::now();
	profile.gpuio = std::chrono::duration_cast<milliseconds>(end - begin);;

	/* Compute D = D^{-1/2} */
	begin = Clock::now();
	log("Computing normalized D");
	compute_d<<<grid, threads>>>(D_device, g.size);
	hipDeviceSynchronize();
	end = Clock::now();
	profile.compute_d = std::chrono::duration_cast<milliseconds>(end - begin);;
	
	/* Compute X = D^{-1/2}AD^{-1/2} */
	log("Computing X");
	begin = Clock::now();
	hipblasSgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
	            A_device,g.size, 
		    D_device, g.size,
		    &bet, 
		    temp_device, g.size);
	hipDeviceSynchronize();	
	hipblasSgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
		    D_device, g.size,
	            temp_device,g.size, 
		    &bet, 
		    X_device, g.size);
	hipDeviceSynchronize();

	end = Clock::now();
	profile.compute_x = std::chrono::duration_cast<milliseconds>(end - begin);;	
	/* Compute S = sum(X^{0}....X^{window_size}) */
	
	hipMemcpy(S_device, X_device, size, hipMemcpyDeviceToDevice);
	hipMemcpy(temp_device, X_device, size, hipMemcpyDeviceToDevice);

	begin = Clock::now();
	for(int i=2;i<=window_size;i++){
		std::cout<<"Computing X^"<<i<<std::endl;
		hipblasSgemm(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, g.size,
		    &al,
		    X_device, g.size,
	            temp_device,g.size, 
		    &bet, 
		    temp1_device, g.size);
		hipDeviceSynchronize();		
		
		// Use cublas addition functions
		hipblasSgeam(handle, 
		    HIPBLAS_OP_N, HIPBLAS_OP_N, 
		    g.size, g.size, 
		    &al,
		    S_device, g.size,
		    &bet, 
	            temp1_device,g.size, 
		    S_device, g.size);
		hipDeviceSynchronize();
		
		hipMemcpy(X_device, temp1_device, size, hipMemcpyDeviceToDevice);
		hipMemset(temp1_device,0,size);
	}

		
	
	// Compute S = S * (vol / (window_size * b))
	transform_s<<<grid,threads>>>(S_device,g.volume, window_size, b, g.size);
        hipDeviceSynchronize();
	end = Clock::now();
	profile.compute_s = std::chrono::duration_cast<milliseconds>(end - begin);;
	
	
	// Compute M = D^{-1/2} * S * D^{-1/2}
	hipMemset(temp_device, 0, size); 
	
	begin = Clock::now();
	log("Computing M");
	hipblasSgemm(handle, 
	    HIPBLAS_OP_N, HIPBLAS_OP_N, 
	    g.size, g.size, g.size,
	    &al,
	    S_device, g.size,
	    D_device,g.size, 
	    &bet, 
	    temp_device, g.size);
        hipDeviceSynchronize();

	hipblasSgemm(handle, 
	    HIPBLAS_OP_N, HIPBLAS_OP_N, 
	    g.size, g.size, g.size,
	    &al,
	    D_device, g.size,
	    temp_device,g.size, 
	    &bet, 
	    M_device, g.size);
        hipDeviceSynchronize();
	
	// Compute M = log(max(Mi,1))
	log("Transforming M");
	transform_m<<<grid,threads>>>(M_device, g.size);
        hipDeviceSynchronize();
	
	end = Clock::now();
	profile.compute_m = std::chrono::duration_cast<milliseconds>(end - begin);;
	
	// Perform SVD on M
	begin = Clock::now();

	hipMemcpy(M, M_device, size, hipMemcpyDeviceToHost);


	log("Printing M_CAP");
	print_matrix(M, g.size);

	log("Creating Eigen matrix");
	Eigen::MatrixXf M_eigen = Eigen::Map<Eigen::MatrixXf>(M, g.size, g.size);

	log("Performing SVD of M");
	REDSVD::RedSVD<Eigen::MatrixXf> svd(M_eigen, dimension);


	log("Sorting singular values");
	 std::vector<unsigned long long int > sorted_indexes(dimension);
	 std::size_t n(0);
	 std::generate(std::begin(sorted_indexes), 
			std::end(sorted_indexes), 
			[&]{ return n++; });

	 std::sort(  std::begin(sorted_indexes), 
	             std::end(sorted_indexes),
	             [&](unsigned long long int  i1, unsigned long long int  i2) { 
				return svd.singularValues()[i1] > svd.singularValues()[i2]; 
			}
		);

	// cout << "\nsort_indexes are \n----------------------------- \n";
	//  for (auto v : sorted_indexes)
	//          std::cout << v << ' ';


	Eigen::PermutationMatrix<Eigen::Dynamic,Eigen::Dynamic,int> perm(dimension);	
	for (unsigned long long int  i = 0; i< dimension; i++)
		perm.indices()[i] = sorted_indexes[i];	

	
	std::cout << "\nSingular Values and U are\n------------------------\n";
	std::cout << "\nSingular Values are " << std::endl << svd.singularValues().transpose() * perm << std::endl ;
	std::cout << "\nComputed U  is " << std::endl << (svd.matrixU() * perm).transpose() << std::endl ;

//	hipsolverDnSgesvd(cusolverH, jobu, jobvt, 
//			g.size, g.size, M_device, g.size, 
//			Si_device, 
//			U_device, g.size, 
//			VT_device, g.size, 
//			d_work, 
//			lwork, 
//			d_rwork, 
//			devInfo); 
	
//        hipDeviceSynchronize();
	log("Moving singular values to host");
	Eigen::Map<Eigen::MatrixXf>(Si, 1, dimension) = svd.singularValues().transpose() * perm;
	Eigen::Map<Eigen::MatrixXf>(U, g.size, dimension) = (svd.matrixU() * perm).transpose();

	log("Moving singular values to device");
	hipMemcpy(Si_device, Si, dimension * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(U_device, U, g.size * dimension * sizeof(float), hipMemcpyHostToDevice);	

	end = Clock::now();
	profile.svd = std::chrono::duration_cast<milliseconds>(end - begin);;	
	
	begin = Clock::now();
	log("Transforming Si");
	sqrt_si<<<grid, threads>>>(Si_device, dimension);	
        hipDeviceSynchronize();

	hipMemcpy(Si, Si_device, dimension * sizeof(float), hipMemcpyDeviceToHost);
	log("Printing sqrt S");
	for(int i=0;i<dimension;i++)
		std::cout<<Si[i]<<" ";

	log("Generating embeddings");
	hipblasSdgmm(handle, 
	    HIPBLAS_SIDE_LEFT, 
	    g.size, dimension,
	    U_device, g.size,
	    Si_device,1, 
	    W_device, g.size);
        hipDeviceSynchronize();
	end = Clock::now();
	profile.emb = std::chrono::duration_cast<milliseconds>(end - begin);;

	hipMemcpy(W, W_device, size, hipMemcpyDeviceToHost);

	write_embeddings("blogcatalog.emb",W, g.size, dimension);	
	write_profile("profile.txt", profile);		
	log("Done");
	/***********
	* Clean up *
	***********/

	free(X);
	free(S);
	free(M);
	
	free(U); 
	free(VT);
	free(Si);
	free(W);

	// DEVICE
	hipFree(D_device);
	hipFree(temp_device); 
	hipFree(temp1_device); 
	hipFree(X_device);
	hipFree(A_device);
	hipFree(S_device);
	hipFree(M_device);

	hipFree(U_device);
	hipFree(VT_device); 
	hipFree(Si_device);
	hipFree(W_device); //auxillary device array

	hipFree(d_work);
	hipFree(d_rwork);
	hipFree(devInfo);

}
